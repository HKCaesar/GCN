#include "hip/hip_runtime.h"
#include <mat.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <matrix.h>
#include <iostream>
#include "hipblas.h"
#include "cokus.cpp"
#include "cuda_util.h"
#include <hip/hip_runtime.h>
using namespace std;

const int KER_NUM = 20;//���������
const int P_NUM = 8;//ÿ�ξ���Ĳ���
const int LEAP = 2;//����
const int GP_NUM = 2;//maxpoolingÿ��ĸ���
const int NEU_NUM1 = 100;
const int NEU_NUM2 = 13;//�������Ԫ����
const int NEIGHBOR = 8;//�����ھӸ���
double LEARN_RATE = 0.008;
const double MIN_ERR = 0.001;
const int VALID_BATCH = 10;

//copy���ݵ�shared memory
__device__ void copy_data_to_shared(double * data, double * data_tmp,int head, int length){
	for(int i=0; i<length; i++){
		data_tmp[i] = data[i+head];
	}

	__syncthreads();
}

//GPU�˸�����
__global__ static void convol(int iter,int i0,double * train,double * kernel,double * re,double * bias,int x,int y,int z,int re_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;//���浱ǰ�̱߳��

	//ÿ���̸߳���һ���������һ��3*3*hight��״ͼ��ľ��
	if (id < KER_NUM){
		extern __shared__ double train_tmp[];
		//__shared__ double train_tmp[9*200];
		int st = i0 * x * y * z;

		copy_data_to_shared(train,train_tmp,st,x*y*z);//����train��shared memory��

		/*double * ker = new double [x*y*P_NUM];//�����Ӧ��kernel���Ĵ���
		for(int i=0; i<x*y*P_NUM; i++){
			ker[i] = kernel[id*x*y*P_NUM + i];
		}*/
		double mid;
		//int i_1=0;
		for(int i=0; i<re_size; i++){
			mid = 0;
			int start = i*x*y*LEAP;//ѵ������ÿ�ξ�������
			for(int j=0; j<x*y*P_NUM; j++){
				mid = mid + train_tmp[start + j]*kernel[id*x*y*P_NUM+j];
			}
			mid = mid + bias[id];
			re[i + id*re_size] = 2/(1+(1/exp(2*mid))) - 1;//�����tanh
		}
		/*for
		}*/
	}
}

//GPU�˽����²���
__global__ static void maxpooling(int iter,double * re,double * mre,int * mre_index,int re_size,int mre_num){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
       	int id = tid + iter * threadNum; 
	
	//int res = re_size, mres = mre_num;
	//extern __shared__ double re_tmp[];
	//copy_data_to_shared(re, re_tmp, 0, re_size*KER_NUM);

	if(id < KER_NUM){
		double mid;
		int mid_index;
		for(int i=0; i<mre_num; i++){
			mid = re[i*GP_NUM + id*re_size];//���ÿ���һ��ֵ
			mid_index = i*GP_NUM + id*re_size;
			for(int j=i*GP_NUM+1; j<(i+1)*GP_NUM && j<re_size; j++){
				if(mid < re[j + id*re_size]){
					mid = re[j + id*re_size];
					mid_index = j+id*re_size;
				}
			}
			mre[i + id * mre_num] = mid;
			mre_index[i + id * mre_num] = mid_index;
		}
	}
}

//ȫ���Ӳ�,ÿ���̸߳���һ����Ԫ�������ļ���
__global__ static void fullconnect(int iter,double * mre,double * omega,double * bias,double * F1,int mre_size){
	int tid = blockIdx.x * blockDim.x +threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;

	if(id < NEU_NUM1){
		//����mre���鵽�����ڴ�
		//__shared__ double mre_tmp[50 * KER_NUM];
	        extern __shared__ double mre_tmp[];	
		copy_data_to_shared(mre,mre_tmp,0,mre_size);
		
		//������Ԫ�����
		double mid=0;
		for(int i=0; i<mre_size; i++){
			mid = mid + omega[id + i*NEU_NUM1] * mre_tmp[i];
		}
		mid = mid + bias[id];
		F1[id] = 2/(1 + 1/exp(mid * 2)) - 1;//�����tanh
	}
}

//����㣬ÿ���̸߳���һ����Ԫ�������ļ���
__global__ static void output(int iter, double * F1, double * omega2, double * bias, double * O2){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;

	if(id < NEU_NUM2){
		//����F1�������ڴ���
		__shared__ double F1_tmp[NEU_NUM1];
		copy_data_to_shared(F1, F1_tmp, 0, NEU_NUM1);
		__shared__ double O2_tmp[NEU_NUM2];

		//������Ԫ�����
		double mid = 0;
		for(int i=0; i<NEU_NUM1; i++){
			mid = mid + omega2[id + i*NEU_NUM2] * F1_tmp[i];
		}
		O2[id] = exp(mid+ bias[id]);
		O2_tmp[id] = O2[id];
		__syncthreads(); //�ȴ������߳̽���Ԫ������������SM

		//����softmax�������������
		int length = NEU_NUM2;//��ǰ��Ҫ�ۼӵ����鳤��
		int offset = (length - 1)/2 +1;//�ۼӵ�ƫ��ֵ
		while(length >= 2)
		{
			if(id + offset < length){
				O2_tmp[id] = O2_tmp[id] + O2_tmp[id + offset];
			}
			offset = (offset - 1)/2 + 1;
			length = (length - 1)/2 + 1;
			__syncthreads();//�ȴ������߳���ɵ�ǰ���ۼ�
		}
		O2[id] = O2[id]/O2_tmp[0];

	}
}

//������ȷ��
double count_err(double * test_labels, double * output, int test_idx)
{
	double right=0;
	double max =0;
	int idx = 0;
	for(int i=0; i<NEU_NUM2; i++){
		if(output[i]>max){
			max = output[i];
			idx = i;
		}
	}
	if((idx+1) == int(test_labels[test_idx]))
		right = 1;
	
	return right;
}

double testint(int test_size, int data_size, double * test_data, double * test_labels, double * kernel, double * omega1, double * omega2, double * bias0, double * bias1, double * bias2)
{
		double * gpu_processed_test;
		double * gpu_kernel;
		double * gpu_omega1;
		double * gpu_omega2;
		double * gpu_bias0;
		double * gpu_bias1;
		double * gpu_bias2;
		double * gpu_re;
		double * gpu_mre;
		double * gpu_mre_index;
		double * gpu_F1;
		double * gpu_O2;
		
			//����ÿ�ξ���Ľ������
		int re_size = 0;
		for (int i=0; i+P_NUM-1<z; i+=LEAP){
			re_size ++;
		}
		int mre_num = (re_size-1)/GP_NUM + 1;
		int mre_size = mre_num * KER_NUM;
		int ome_num1 = mre_num * KER_NUM * NEU_NUM1;//��һ�����������Ȩ�ظ���
		int ome_num2 = NEU_NUM1 * NEU_NUM2;//������Ȩ�ظ���	
		
		SAFE_CALL(hipMalloc((void **) &gpu_processed_test, sizeof(double) * data_size));
		SAFE_CALL(hipMalloc((void **) &gpu_kernel,sizeof(double) * (NEIGHBOR+1) * P_NUM * KER_NUM));
		SAFE_CALL(hipMalloc((void **) &gpu_omega1, sizeof(double) * ome_num1));//��һ�����������Ȩ�أ������Դ�
		SAFE_CALL(hipMalloc((void **) &gpu_omega2, sizeof(double) * ome_num2));//������Ȩ�أ������Դ�
		SAFE_CALL(hipMalloc((void **) &gpu_bias0, sizeof(double) * KER_NUM));//�����ƫ��ֵ
		SAFE_CALL(hipMalloc((void **) &gpu_bias1, sizeof(double) * NEU_NUM1));//ȫ���Ӳ�ƫ��ֵ
		SAFE_CALL(hipMalloc((void **) &gpu_bias2, sizeof(double) * NEU_NUM2));//�����ƫ��
		SAFE_CALL(hipMalloc((void **) &gpu_re,sizeof(double) * re_size * KER_NUM));
		SAFE_CALL(hipMalloc((void **) &gpu_mre, sizeof(double) * mre_num * KER_NUM));//maxpooling�������gpu_mre�������Դ�
		SAFE_CALL(hipMalloc((void **) &gpu_mre_index, sizeof(int) * mre_num * KER_NUM));//Ϊmaxpooling�����ֵ���������Դ�
		SAFE_CALL(hipMalloc((void **) &gpu_F1, sizeof(double) * NEU_NUM1));//��һ�����������������Դ�
		SAFE_CALL(hipMalloc((void **) &gpu_O2, sizeof(double) * NEU_NUM2));//�����Ľ��
		
		SAFE_CALL(hipMemcpy(gpu_processed_test,test_data,sizeof(double) * (NEIGHBOR+1) * data_size, hipMemcpyHostToDevice));
		SAFE_CALL(hipMemcpy(gpu_kernel,kernel,sizeof(double) * (NEIGHBOR+1) * P_NUM * KER_NUM,hipMemcpyHostToDevice));
		SAFE_CALL(hipMemcpy(gpu_omega1, omega1, sizeof(double) * ome_num1, hipMemcpyHostToDevice));//���Ƴ�ʼȨ�ص�GPU��
		SAFE_CALL(hipMemcpy(gpu_omega2, omega2, sizeof(double) * ome_num2, hipMemcpyHostToDevice));
		SAFE_CALL(hipMemcpy(gpu_bias0, bias0, sizeof(double) * KER_NUM, hipMemcpyHostToDevice));
		SAFE_CALL(hipMemcpy(gpu_bias1, bias1, sizeof(double) * NEU_NUM1, hipMemcpyHostToDevice));//����ƫ��ֵ���Դ�
		SAFE_CALL(hipMemcpy(gpu_bias2, bias2, sizeof(double) * NEU_NUM2, hipMemcpyHostToDevice));
		
		double right = 0;
		double count0 = 0;
		for (int i1=0; i1<test_size; i1++){
			int iter = 0;
			convol<<<1,KER_NUM,(NEIGHBOR+1)*z*sizeof(double)>>>(iter,i1,gpu_processed_test,gpu_kernel,gpu_re,gpu_bias0,3,3,z,re_size);
			hipDeviceSynchronize();

			maxpooling<<<1,KER_NUM>>>(iter,gpu_re,gpu_mre,gpu_mre_index,re_size,mre_num);
			hipDeviceSynchronize();

			fullconnect<<<1,NEU_NUM1,mre_size * sizeof(double)>>>(iter,gpu_mre,gpu_omega1,gpu_bias1,gpu_F1,mre_size);
			hipDeviceSynchronize();

			output<<<1,NEU_NUM2>>>(iter,gpu_F1,gpu_omega2,gpu_bias2,gpu_O2);
			hipDeviceSynchronize();

			SAFE_CALL(hipMemcpy(O2, gpu_O2, sizeof(double) * NEU_NUM2, hipMemcpyDeviceToHost));
			hipDeviceSynchronize();

			//fprintf(stdout,"\n");
			right = count_err(test_labels, O2, i1);
			count0 = count0 + right;
		}
		
		return count0/test_size;
}
int main(int argc, char * argv[])
{
	clock_t start,end;

	double * kernel,* omega1, * omega2, * bias0, * bias1, * bias2;
	if(argc!=3){
		fprintf(stderr, "3 input arguments required!");
	}
	MATFile * datamat = matOpen(argv[1], "r");
	mxArray * ker = matGetVariable(datamat,"kernel");
	mxArray * ome1 = matGetVariable(datamat,"omega1");
	mxArray * ome2 = matGetVariable(datamat,"omega2");
	mxArray * b0 = matGetVariable(datamat,"bias0");
	mxArray * b1 = matGetVariable(datamat,"bias1");
	mxArray * b2 = matGetVariable(datamat,"bias2");

	kernel = (double*)mxGetData(ker);
	omega1 = (double*)mxGetData(ome1);
	omega2 = (double*)mxGetData(ome2);
	bias0 = (double*)mxGetData(b0);
	bias1 = (double*)mxGetData(b1);
	bias2 = (double*)mxGetData(b2);
	matClose(datamat);
	
	double * test_data, * test_labels;
	MATFile * testmat = matOpen(argv[2], "r");
	mxArray * data = matGetVariable(testmat,"data");
	mxArray * labels = matGetVariable(testmat,"labels");
	
	test_data = (double*)mxGetData(data);
	test_labels	= (double*)mxGetData(labels);
	const mwSize  * dim0, *dim1;
	dim0 = mxGetDimensions(labels);//��ȡ���Լ�����
	dim1 = mxGetDimensions(data);//��ȡ���Լ���ģ
	matClose(testmat);

	double corr = testing(dim0[0],dim1[0] * dim1[1] * dim1[2],test_data,test_labels,kernel,omega1,omega2,bias0,bias1,bias2);	
}