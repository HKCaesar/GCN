#include "hip/hip_runtime.h"
#include <mat.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <matrix.h>
#include <iostream>
#include "error_util.h"
#include <hip/hip_runtime.h>
#include <algorithm>
using namespace std;

const int FILTER_NUM = 20;
const int COV_LEN = 19;
const int STRIDE = 2;
const int POOLONG_LEN = 2;
const int NEU_NUM1 = 100;
const int NEU_NUM2 = 13;
const int NEIGHBOR = 8;
double learning_rate = 0.5;
const double MIN_ERR = 0.0001;
const int VALID_BATCH = 5;
const int DATA_BATCH = 100;

//Initialize CUDA
bool InitCUDA(){
    int count;
    hipGetDeviceCount(&count);
    if(count==0){
        fprintf(stderr,"There is no device.\n");
        return false;
    }
    int i;
    for (i =0; i<count;i++){
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop,i)==hipSuccess){
            if(prop.major>=1){       
                break;
            }
        }
    }
    if( i == count ) {
        fprintf(stderr,"There is no device supporting CUDA.\n");
        return false;
    }
    // hipSetDevice(i);
    return true;
}

template<typename T>
struct Tensor{
    int length;
    T * data_h;
    T * data_d;

    Tensor();
};

template<typename T>
Tensor<T>::Tensor()
{
    length = 0;
    data_h = NULL;
    data_d = NULL;
}

struct DataLayer{
    Tensor<double> input;
    Tensor<int> labels;

    DataLayer(int input_size, int labels_size);
    ~DataLayer();
};

DataLayer::DataLayer(int input_size, int labels_size)
{
    input.length = input_size;
    labels.length = labels_size;
}

DataLayer::~DataLayer()
{
    if(input.data_h != NULL)
        delete [] input.data_h;
    if(labels.data_h != NULL)
        delete [] labels.data_h;
    if(input.data_d != NULL)
        hipFree(input.data_d);
    if(labels.data_d != NULL)
        hipFree(labels.data_d);
}

struct Layer{
    Tensor<double> input;
    Tensor<double> output;
    Tensor<double> weights;
    Tensor<double> bias;
    Tensor<double> deltaW;
    Tensor<double> deltaB;

    Layer(int input_size, int weights_size, int bias_size, int output_size, int batch_size, bool isMaxpooling, bool copyback);
    ~Layer();

private:
    void allocMemcpyCuda(int size, double ** data_h, double ** data_d, bool isMallochost, bool isInitalize);
};

void Layer::allocMemcpyCuda(int size, double **data_h, double **data_d, bool isMallochost, bool isInitalize)
{
    size_t sizeBytes = size * sizeof(double);
    checkCudaErrors(hipMalloc(data_d, sizeBytes));

    if (isMallochost)
    {
        *data_h = new double [size];

        if (isInitalize)
        {
            for (int i = 0; i < size; i ++)
            {
                data_h[0][i] = (2 * (rand()/double(RAND_MAX)) - 1) / 50;
            }
            checkCudaErrors(hipMemcpy(*data_d, *data_h, sizeBytes, hipMemcpyHostToDevice));
        }
    }
}

Layer::Layer (int input_size, int weights_size, int bias_size, int output_size, int batch_size, bool isMaxpooling, bool copyback)
{
    input.length = input_size * batch_size;
    weights.length = weights_size;
    deltaW.length = weights_size * batch_size;
    output.length = output_size * batch_size;
    bias.length = bias_size;
    deltaB.length = bias_size * batch_size;

    if ( isMaxpooling )
        bias.length = bias_size * batch_size;

    allocMemcpyCuda(input.length, &input.data_h, &input.data_d, false, false);
    allocMemcpyCuda(weights.length, &weights.data_h, &weights.data_d, true, true);
    allocMemcpyCuda(bias.length, &bias.data_h, &bias.data_d, true, true);
    allocMemcpyCuda(output.length, &output.data_h, &output.data_d, copyback, false);
    allocMemcpyCuda(deltaB.length, &deltaB.data_h, &deltaB.data_d, false, false);
    allocMemcpyCuda(deltaW.length, &deltaW.data_h, &deltaW.data_d, false, false);

    if ( isMaxpooling )
        checkCudaErrors(hipMemset(deltaW.data_d, 0, sizeof(double) * deltaW.length));
}

Layer::~Layer ()
{
    if ( input.data_h != NULL )
        delete [] input.data_h;
    if ( weights.data_h != NULL )
        delete [] weights.data_h;
    if ( output.data_h != NULL )
        delete [] output.data_h;
    if ( bias.data_h != NULL )
        delete [] bias.data_h;
    if ( deltaW.data_h != NULL )
        delete [] deltaW.data_h;
    if ( deltaB.data_h != NULL )
        delete [] deltaB.data_h;
    if ( input.data_d != NULL )
        hipFree(input.data_d);
    if ( output.data_d != NULL )
        hipFree(output.data_d);
    if ( weights.data_d != NULL )
        hipFree(weights.data_d);
    if ( bias.data_d != NULL )
        hipFree(bias.data_d);
    if ( deltaW.data_d != NULL )
        hipFree(deltaW.data_d);
    if ( deltaB.data_d != NULL )
        hipFree(deltaB.data_d);
}

// copy data to shared memory
__device__ void copy_data_to_shared( double * data, double * data_tmp, int tid, int offset, int head, int length )
{
    for(size_t i = tid * offset; i < (tid + 1) * offset && (i < length); i++)
    {
        data_tmp[i] = data[i + head];
    }
    __syncthreads();

}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// forward propagation kernels
// forward convolution
__global__ static void convolution( int data_id,
                                    int batch_id,
                                    int perLayerSize,
                                    int cube_len,
                                    int conv_len,
                                    int filter_num,
                                    int stride,
                                    double * input, 
                                    double * filters, 
                                    double * bias,
                                    double * output )
{

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int output_size = (cube_len - conv_len - 1) / stride + 1;

    if ( tid < output_size && bid < filter_num )
    {
        int cube_size = cube_len * perLayerSize;
        extern __shared__ double tmp[];
        int head = data_id * cube_size;
        int length = cube_size;
        int offset = (length - 1) / output_size + 1;
        copy_data_to_shared(input, tmp, tid, offset, head, length);
        
        int filterSize = conv_len * perLayerSize;
        head = bid * filterSize;
        length = filterSize;
        offset = (length - 1) / output_size + 1;
        copy_data_to_shared(filters, tmp + cube_size, tid, offset, head, length);
        __syncthreads();

        double mid = 0;
        for(int i = 0; i < filterSize; i++){
            mid = mid + tmp[i + cube_size] * tmp[tid * perLayerSize * stride + i];
        }
        mid = mid + bias[bid];

        output[tid + bid * output_size + batch_id * output_size * filter_num] = 2 / (1 + (1 / exp(2 * mid))) - 1;
    }
}

// forward maxpooling
__global__ static void maxpooling( int batch_id,
                                   int input_size,
                                   int pooling_size,
                                   int group_num,
                                   double * input,
                                   double * output, 
                                   double * output_index )
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int output_size = (input_size - 1) / pooling_size + 1;

    if ( tid < output_size && bid < group_num )
    {
        double max;
        int max_index, head, tail;
        head = tid * pooling_size + bid * input_size + batch_id * input_size * group_num;
        tail = (tid + 1) * pooling_size + bid * input_size + batch_id * input_size * group_num;
        max = input[head];
        max_index = head;
        for ( int i = head; i < tail && (i < (bid + 1) * input_size + batch_id * input_size * group_num); i ++ )
        {
            if(max < input[i]){
                max = input[i];
                max_index=i;
            }
        }

        output[tid + bid * output_size + batch_id * output_size * group_num] = max;
        output_index[tid + bid * output_size + batch_id * output_size * group_num] = max_index;
    }
}

// forward fully connection
__global__ static void fully_connect( int batch_id,
                                      int input_size,
                                      int neuron_num,
                                      double * input,
                                      double * weights,
                                      double * bias,
                                      double * output )
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    if ( tid < input_size && bid < neuron_num )
    {
        extern __shared__ double tmp[];
        tmp[tid] = weights[bid + tid * neuron_num] * input[tid + batch_id * input_size];
        __syncthreads();

        int length = input_size;
        int offset = (length - 1) / 2 + 1;

        while ( length >= 2 )
        {
            if( tid + offset < length )
            {
                tmp[tid] = tmp[tid] + tmp[tid + offset];
            }
            offset = (offset - 1)/2 + 1;
            length = (length - 1)/2 + 1;
            __syncthreads();
        }

        if ( tid < 1 )
            output[bid + batch_id * neuron_num] = 2/(1 + 1 / exp((tmp[0] + bias[bid]) * 2)) - 1;
    }
}

// forward output
__global__ static void output_and_dvalue( int data_id,
                                          int batch_id,
                                          int input_size,
                                          int neuron_num, 
                                          bool isBackwardPropagation,
                                          double * input, 
                                          double * weights, 
                                          double * bias, 
                                          double * output,
                                          int * labels,
                                          double * dValue )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( tid < neuron_num )
    {
        // copy to shared memory
        extern __shared__ double tmp[];
        int offset = (input_size - 1) / neuron_num + 1;
        copy_data_to_shared(input, tmp, tid, offset, batch_id * input_size, input_size);
        __syncthreads();

        double mid = 0;
        for ( int i = 0; i < input_size; i ++ ) {
            mid = mid + weights[tid + i * neuron_num] * tmp[i];
        }

        tmp[tid + input_size] = exp(mid + bias[tid]);
        __syncthreads(); 

        int length = neuron_num;
        offset = (length - 1) / 2 + 1;
        while(length >= 2)
        {
            if(tid + offset < length){
                tmp[tid + input_size] = tmp[tid + input_size] + tmp[tid + input_size + offset];
            }
            offset = (offset - 1) / 2 + 1;
            length = (length - 1) / 2 + 1;
            __syncthreads();
        }

        output[tid + batch_id * neuron_num] = exp(mid + bias[tid]) / tmp[input_size];
        
        if ( isBackwardPropagation )
            dValue[tid + batch_id * neuron_num] = (output[tid + batch_id * neuron_num] - labels[tid + data_id * neuron_num]) / neuron_num;
    }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// backward propagation kernels
// output layer
/*__global__ static void bp_output( int batch_id,
                                         int input_size,
                                         int output_size, 
                                         double * weights, 
                                         double * deltaB, 
                                         double * deltaW,
                                         double * data, 
                                         double * fol_deltaZ )
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    if ( tid < output_size && bid < input_size ) {
        extern __shared__ double delta_A[];
        delta_A[tid] = weights[tid + bid * output_size] * 
                       deltaB[tid + batch_id * output_size];
        __syncthreads();

        deltaW[tid + 
               bid * output_size + 
               batch_id * input_size * output_size] = data[bid + batch_id * input_size] * 
                                                      deltaB[tid + batch_id * output_size]; 

        int length = output_size;
        int offset = (length - 1)/2 + 1;
        while ( length >= 2 )
        {
            if(tid + offset < length){
                delta_A[tid] = delta_A[tid] + delta_A[tid + offset];
            }
            length = (length - 1)/2 + 1;
            offset = (offset - 1)/2 + 1;
            __syncthreads();
        }

        if ( tid < 1 )
            fol_deltaZ[bid + batch_id * input_size] = delta_A[0] * 
                                                      (1 + data[bid + batch_id * input_size]) *
                                                      (1 - data[bid + batch_id * input_size]);
    }
}*/

// fully_connect layer
__global__ static void bp_fully_connect( int batch_id, 
                                      int input_size, 
                                      int output_size,
                                      double * weights,
                                      double * deltaB,
                                      double * deltaW,
                                      double * data,
                                      double * fol_deltaZ )
                                    
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    if( tid < output_size && bid < input_size )
    {
        extern __shared__ double mid[];
        mid[tid] = weights[tid + bid * output_size] * deltaB[tid + batch_id * output_size];
        __syncthreads();

        int length = output_size;
        int offset = (length - 1)/2 + 1;
        while ( length >= 2 ) {
            if(tid + offset < length){
                mid[tid] = mid[tid] + mid[tid+offset];
            }
            length = (length - 1)/2 + 1;
            offset = (offset - 1)/2 + 1;
            __syncthreads();
        }

        double data_tmp = data[bid + batch_id * input_size];

        deltaW[tid + bid * output_size + batch_id * input_size * output_size] = data_tmp * deltaB[tid + batch_id * output_size];
        if(tid < 1)
            fol_deltaZ[bid + batch_id * input_size] = mid[0] * (1 + data_tmp) * (1 - data_tmp);
    }
}

// maxpooling
__global__ static void bp_maxpooling( int batch_id,
                                      int input_size,
                                      int output_size,
                                      double * bias,
                                      double * deltaB,
                                      double * fol_deltaZ )
{
    int tid = threadIdx.x;

    if ( tid < output_size )
    {
        int idx = (int)bias[tid + batch_id * output_size];

        fol_deltaZ[idx] = deltaB[tid + batch_id * output_size];
    }
}

// convolutional layer
__global__ static void bp_convolution( int data_id, 
                                       int batch_id,  
                                       int stride, 
                                       int perLayerSize,
                                       int cube_len,
                                       int filter_size,
                                       int filter_num,
                                       int output_size,
                                       double * pre_deltaB, 
                                       double * deltaW, 
                                       double * deltaB, 
                                       double * data,
                                       double * output )
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int re_size = output_size / filter_num;
    if ( tid < filter_size && bid < filter_num)
    {
        int cube_size = cube_len * perLayerSize;
        int head = data_id * cube_size;
        int length = cube_size;
        int offset = (length - 1)/filter_size + 1;
        extern __shared__ double data_tmp[];
        copy_data_to_shared(data, data_tmp, tid, offset, head, length);
        __syncthreads();

        double mid0 = 0, mid1 = 0;
        for( int i = 0; i < re_size; i ++ ) {
            mid0 = mid0 + pre_deltaB[i + bid * re_size + batch_id * output_size] * data_tmp[tid + i * perLayerSize * stride];
            mid1 = mid1 + pre_deltaB[i + bid * re_size + batch_id * output_size] * (1 + output[i + bid * re_size + batch_id * output_size])
                                                                                 * (1 - output[i + bid * re_size + batch_id * output_size]);
        }

        deltaW[tid + bid * filter_size + batch_id * filter_size * filter_num] = mid0 / re_size;
        
        if ( tid < 1 )
            deltaB[bid + batch_id * filter_num] = mid1 / re_size;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// update params kernels
__global__ static void update_params_row( int batch_size, 
                                          int input_size,
                                          int output_size,
                                          double lr, 
                                          double * weights, 
                                          double * deltaW,
                                          double * bias,
                                          double * deltaB )
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    if ( tid < output_size && bid < input_size )
    {
        double tmp0 = 0, tmp1 = 0;
        for ( int i = 0; i < batch_size; i ++ )
        {
            tmp0 = tmp0 + deltaW[tid + bid * output_size + i * input_size * output_size]; 
        }

        weights[tid + bid * output_size] = weights[tid + bid * output_size] - lr * tmp0 / batch_size;

        if ( bid < 1 ) {
            for ( int j = 0; j < batch_size; j++ )
                tmp1 = tmp1 + deltaB[tid + j * output_size];

            bias[tid] = bias[tid] - lr * tmp1 / batch_size;
        }
    }
}


// convolution layer
__global__ static void update_params_col( int batch_size, 
                                          int filter_size,
                                          int filter_num,
                                          double lr, 
                                          double * filters,
                                          double * deltaW,
                                          double * bias,  
                                          double * deltaB )
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    if ( tid < filter_size && bid < filter_num )
    {
        double tmp0 = 0, tmp1 = 0;
        for( int i = 0; i < batch_size; i ++ ) 
        {
            tmp0 = tmp0 + deltaW[tid + bid * filter_size + i * filter_size * filter_num];
            tmp1 = tmp1 + deltaB[bid + i * filter_num];
        }
        filters[tid + bid * filter_size] = filters[tid + bid * filter_size] - lr * tmp0 / batch_size;
        
        if ( tid < 1 ) {
            bias[bid] = bias[bid] - lr * tmp1 / batch_size;
        }    
    }
}

__global__ static void loss_function( int batch_id, 
                                      int batch_size, 
                                      int output_size,
                                      double * output, 
                                      int * labels, 
                                      double * loss_values)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //double sum = 0.0;
    //double tmp = 0.0;
    if ( tid < batch_size )
    {
        double tmp = 0.0;
        /*int count_pos = 0;
        int count_neg = 0;
        double temp_loss_pos = 0;
        double temp_loss_neg = 0;*/

        for ( size_t i = 0; i < output_size; i ++ ) {
            tmp = tmp + labels[i + (batch_id * batch_size + tid) * output_size] * log(output[i + tid * output_size]) +
                  (1 - labels[i + (batch_id * batch_size + tid) * output_size]) * log(1 - output[i + tid * output_size]);
            /*int idx = i + (batch_id * batch_size + tid) * output_size;
            if ( labels[idx] == 1 ) {
                count_pos = count_pos + 1;
                temp_loss_pos -= output[i + tid * output_size] * (labels[idx] - (output[i + tid * output_size] >= 0)) - 
                    log(1 + exp(output[i + tid * output_size] - 2 * output[i + tid * output_size] * (output[i + tid * output_size >= 0])));
            }
            else if ( labels[idx] == 0 ) {
                count_neg ++;
                temp_loss_neg -= output[i + tid * output_size] * (labels[idx] - (output[i + tid * output_size] >= 0)) -
                    log(i + exp(output[i + tid * output_size] - 2 * output[i + tid * output_size] * (output[i + tid * output_size] >= 0)));
            }*/
        }

        loss_values[tid] = /*(temp_loss_pos * count_neg / output_size) * 1 + (temp_loss_neg * count_pos / output_size);*/ -tmp / output_size;
    }
}

//preprocessing
__global__ static void preprocessing(int iter, double * data, int * train_index, double * processed_data, int x, int y, int z, int train_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int threadNum = blockDim.x * gridDim.x;
    int id = tid + iter * threadNum;

    if  ( id < train_size ) {
        int idx = id * (NEIGHBOR+1) * z;
        int i, j;
        for ( i = 0; i < z; i ++ ) {
            for ( j = 0; j < (NEIGHBOR + 1); j ++ ) {
                processed_data[idx] = data[train_index[j + id * (NEIGHBOR + 1)] + i * x * y];
                idx = idx + 1;    
            }
        }
    }
}

double lossfunction(double * output, int * labels, int idx)
{
    double l = 0;
    for ( int i = 0; i < NEU_NUM2; i ++ )
    {
        l += labels[i + idx * NEU_NUM2] * log(output[i]) + (1 - labels[i + idx * NEU_NUM2]) * log(1 - output[i]);
    }
    l = - l / NEU_NUM2;
    
    return l;
}

// compute correct rate
double count_err(double * test_labels, double * output, int test_idx)
{
    double right=0;
    double max =0;
    int idx = 0;
    
    for ( int i = 0; i < NEU_NUM2; i ++ )
    {
        if ( output[i] > max )
        {
            max = output[i];
            idx = i;
        }
    }
    if ( (idx + 1) == int(test_labels[test_idx]) )
        right = 1;
    
    return right;
}

// Insert current loss value to the queue
void insert_line(double * a, double b)
{
    for ( int i = 1; i < VALID_BATCH; i ++ ) {
        a[i - 1] = a[i];
    }
    a[VALID_BATCH - 1] = b;
}

// shuffle
void shuffle(int * data, int * labels, int dim_row, int width)
{
    int index,  i;
    int temp;
    double tmp;
    srand(time(NULL));
    for ( i = 0; i < width; i ++ )
    {
        index = rand() % (width - i) + i;
        if ( index != i ) {
            for ( int j = 0; j < dim_row; j ++ )
            {
                temp = data[j + i * dim_row];
                data[j + i * dim_row] = data[j + index * dim_row];
                data[j + index * dim_row] = temp;
            }

            for ( int j = 0; j < NEU_NUM2; j ++ ) 
            {
                tmp = labels[j + i * NEU_NUM2];
                labels[j + i * NEU_NUM2] = labels[j + index * NEU_NUM2];
                labels[j + index * NEU_NUM2] = tmp;
            }
        }
    }
}

double training(double * data, double * labels, int x, int y, int z)
{
    clock_t start, end;
    start = clock();    
    double * gpu_data;
    double * gpu_processed_train;
    double * gpu_processed_test;
    int * gpu_train_index;
    int * gpu_test_index;
    int * gpu_processed_labels;

    //preprocessing
    int data_size = 0;
    int * data_index = new int [x * y];
    for ( int i = 0; i < x * y; i ++ ) 
    {
        if ( labels[i] != 0 ) {
            data_index[data_size] = i;
            data_size ++;
        }
    }
    int test_size = (data_size - 1) / 5 + 1;
    int train_size = data_size - test_size;
    int * train_index = new int [train_size * (NEIGHBOR + 1)];
    int * test_index = new int [test_size * (NEIGHBOR + 1)];

    int * processed_labels = new int [train_size * NEU_NUM2]();
    double * test_labels = new double [test_size]();

    int tr=0, te=0;
    for (int i = 0; i < data_size; i ++ ) {
        if (i % 5 != 0 ) {
            train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1)] = data_index[i]; //index of current labeled pixel
            if ( NEIGHBOR == 4 )
            {
                train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1) - 1] = data_index[i] - 1;
                train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1) + 1] = data_index[i] + 1;
                train_index[0 + tr * (NEIGHBOR + 1)] = data_index[i] - x;
                train_index[NEIGHBOR + tr * (NEIGHBOR + 1)] = data_index[i] + x;
                
                if ( (data_index[i] % x) == 0 ) { //first row
                    train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1) - 1] = train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1) + 1];
                }
                if ( (data_index[i] % x) == (x-1) ) { //last row
                    train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1) + 1] = train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1) - 1];
                }
                if ( (data_index[i] / x) == 0 ) { //first column
                    train_index[0 + tr * (NEIGHBOR + 1)] = train_index[NEIGHBOR + tr * (NEIGHBOR + 1)];
                }
                if ( (data_index[i] / x) == (y - 1) ) { //last column
                    train_index[NEIGHBOR + tr * (NEIGHBOR + 1)] = train_index[0 + tr * (NEIGHBOR + 1)];
                }
            }
            if ( NEIGHBOR == 8 )
            {
                train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1) - 1] = data_index[i] - 1;
                train_index[(NEIGHBOR / 2) + tr * (NEIGHBOR + 1) + 1] = data_index[i] + 1;
                for ( int j0 = 0; j0 < 3; j0 ++ ) {
                    train_index[j0 + tr * (NEIGHBOR + 1)] = data_index[i] - 1 - x + j0;
                    train_index[j0 + 6 + tr * (NEIGHBOR + 1)] = data_index[i] - 1 + x + j0;
                }

                if ( (data_index[i] % x) == 0 ) { //first row
                    for ( int j = 0; j < 3; j ++ )
                        train_index[j * 3 + tr * (NEIGHBOR + 1)] = train_index[j * 3 + 2 + tr * (NEIGHBOR + 1)];
                }
                if ( (data_index[i] % x) == (x - 1) ) { //last row
                    for ( int j = 0; j < 3; j ++ )
                        train_index[j * 3 + 2 + tr * (NEIGHBOR + 1)] = train_index[j * 3 + tr * (NEIGHBOR + 1)];
                }
                if ( (data_index[i] / x) == 0 ) { //first column
                    for ( int j = 0; j < 3; j ++ )
                        train_index[j + tr * (NEIGHBOR + 1)] = train_index[j + 6 + tr * (NEIGHBOR + 1)];
                }
                if ( (data_index[i] / x) == (y - 1) ) { //last column
                    for ( int j = 0; j < 3; j ++ )
                        train_index[j + 6 + tr * (NEIGHBOR + 1)] = train_index[j + tr * (NEIGHBOR + 1)];
                }
            }

            int mid = int(labels[data_index[i]]) - 1 + tr * NEU_NUM2;
            processed_labels[mid] = 1;
            tr = tr + 1;
        }
        if ( i % 5 == 0) {
            test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1)] = data_index[i]; //index of current labeled pixel
            if ( NEIGHBOR == 4 )
            {
                test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1) - 1] = data_index[i] - 1;
                test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1) + 1] = data_index[i] + 1;
                test_index[0 + te * (NEIGHBOR + 1)] = data_index[i] - x;
                test_index[NEIGHBOR + te * (NEIGHBOR + 1)] = data_index[i] + x;

                if ( (data_index[i] % x) == 0 ) { //first row
                    test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1) - 1] = test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1) + 1];
                }
                if ( (data_index[i] % x) == (x - 1) ) { //last row
                    test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1) + 1] = test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1) - 1];
                }
                if ( (data_index[i] / x) == 0 ) { //first column
                    test_index[0 + te * (NEIGHBOR + 1)] = test_index[NEIGHBOR+ te * (NEIGHBOR + 1)];
                }
                if ( (data_index[i] / x) == (y - 1) ) { //last column
                    test_index[NEIGHBOR+ te * (NEIGHBOR+1)] = test_index[0 + te * (NEIGHBOR+1)];
                }
            }
            if ( NEIGHBOR == 8 )
            {
                test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1) - 1] = data_index[i] - 1;
                test_index[(NEIGHBOR / 2) + te * (NEIGHBOR + 1) + 1] = data_index[i] + 1;
                for ( int j0 = 0; j0 < 3; j0 ++ ) {
                    test_index[j0 + te * (NEIGHBOR + 1)] = data_index[i] - 1 - x + j0;
                    test_index[j0 + 6 + te * (NEIGHBOR + 1)] = data_index[i] - 1 + x + j0;
                }

                if ( (data_index[i] % x) == 0 ) { //first row
                    for ( int j = 0; j < 3; j ++ )
                        test_index[j * 3 + te * (NEIGHBOR + 1)] = test_index[j * 3 + 2 + te * (NEIGHBOR + 1)];
                }
                if ( (data_index[i] % x) == (x - 1) ) { //last row
                    for ( int j = 0; j < 3; j ++ )
                        test_index[j * 3 + 2 + te * (NEIGHBOR + 1)] = test_index[j * 3 + te * (NEIGHBOR + 1)];
                }
                if ( (data_index[i] / x) == 0 ) { //first column
                    for ( int j = 0; j < 3; j++ )
                        test_index[j + te * (NEIGHBOR + 1)] = test_index[j + 6 + te * (NEIGHBOR + 1)];
                }
                if ( (data_index[i] / x) == (y - 1) ) { //last column
                    for ( int j = 0; j < 3; j ++ )
                        test_index[j + 6  + te * (NEIGHBOR + 1)] = test_index[j + te * (NEIGHBOR + 1)];
                }
            }

            test_labels[te] = labels[data_index[i]];
            te = te + 1;
        }
    }

    shuffle(train_index, processed_labels, (NEIGHBOR + 1), train_size); //shuffle the samples in training set

    //malloc GPU memory, copy data to GPU
    checkCudaErrors(hipMalloc((void **) &gpu_data, sizeof(double) * x * y * z));
    checkCudaErrors(hipMemcpy(gpu_data, data, sizeof(double)* x * y * z, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void **) &gpu_train_index, sizeof(int) * train_size * (NEIGHBOR+1)));
    checkCudaErrors(hipMemcpy(gpu_train_index, train_index, sizeof(int) * train_size * (NEIGHBOR+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &gpu_test_index, sizeof(int) * test_size * (NEIGHBOR+1)));
    checkCudaErrors(hipMemcpy(gpu_test_index, test_index, sizeof(int) * test_size * (NEIGHBOR+1), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void **) &gpu_processed_test, sizeof(double) * test_size * (NEIGHBOR+1) * z));
    checkCudaErrors(hipMalloc((void **) &gpu_processed_train, sizeof(double) * train_size * (NEIGHBOR+1) *z));

    delete [] data_index;
    delete [] train_index;
    delete [] test_index;

    int gridsize = 64;
    int blocksize = 512;
    int iter=0;

    preprocessing<<<gridsize,blocksize>>>(iter, gpu_data, gpu_train_index, gpu_processed_train, x, y, z, train_size);
    preprocessing<<<gridsize,blocksize>>>(iter, gpu_data, gpu_test_index, gpu_processed_test, x, y, z, test_size);

    //hipDeviceSynchronize();
    end = clock();
    float tt = float(end - start);
    fprintf(stdout,"[Samples prepared with %d Nearest-Neighbor-Pixels Strategy] Proportion of Training Samples: %d%%  Execution time: %.3f sec\n", 
            NEIGHBOR, 80, tt/CLOCKS_PER_SEC);

    checkCudaErrors(hipFree(gpu_data));
    checkCudaErrors(hipFree(gpu_train_index));
    checkCudaErrors(hipFree(gpu_test_index));
    
    // calculate length of convolutional results
    int re_size = 0;
    for ( int i = 0; i + COV_LEN < z; i += STRIDE )
    {
        re_size ++;
    }
    int mre_size = (re_size-1) / POOLONG_LEN + 1;
    int pooling_input_length = re_size * FILTER_NUM;
    int pooling_output_length = mre_size * FILTER_NUM;
    int ful_weights_size = pooling_output_length * NEU_NUM1;// Weights in full connection layer
    int out_weights_size = NEU_NUM1 * NEU_NUM2;// Weights in output layer
    int filter_size = (NEIGHBOR + 1) * COV_LEN;
    int cube_size = (NEIGHBOR + 1) * z;
    
    double * gpu_loss_values;

    // copy labels to GPU
    checkCudaErrors(hipMalloc((void**) &gpu_processed_labels, sizeof(int) * train_size * NEU_NUM2));
    checkCudaErrors(hipMemcpy(gpu_processed_labels, processed_labels, sizeof(int) * train_size * NEU_NUM2,hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &gpu_loss_values, sizeof(double) * DATA_BATCH));

    delete [] processed_labels;
    
    double loss;
    double * logloss = new double [1000]();
    double * loss_values = new double [DATA_BATCH];
    double * correct_rate = new double [VALID_BATCH];
    for ( int i = 0; i < VALID_BATCH; i ++ ) {
        correct_rate[i] = 100;
    }

    double * O2 = new double [NEU_NUM2 * DATA_BATCH]();
    double cur_min = 100;
    int count = 1;
    int batch_size = 0;
    int batch_num = train_size / DATA_BATCH;
    // cout << "batch_num = " << batch_num <<endl;

    start = clock();

    DataLayer dataLayer(train_size * cube_size, train_size * NEU_NUM2);
    dataLayer.input.data_d = gpu_processed_train;
    dataLayer.labels.data_d = gpu_processed_labels;

    Layer conv( cube_size, filter_size * FILTER_NUM, FILTER_NUM, pooling_input_length, DATA_BATCH, false, false);

    Layer pooling(pooling_input_length, pooling_input_length, pooling_output_length, pooling_output_length, DATA_BATCH, true, false);
    
    Layer fulconnect(pooling_output_length, pooling_output_length * NEU_NUM1, NEU_NUM1, NEU_NUM1, DATA_BATCH, false, false);

    Layer out(NEU_NUM1, NEU_NUM1 * NEU_NUM2, NEU_NUM2, NEU_NUM2, DATA_BATCH, false, true);

    hipDeviceSynchronize();
    int max_iter = 300;
    // double decay_ratio = 0.95;
    // double ra = 0.0001;
    fprintf(stdout, "[Cube CNN training with MBGD algo.  BatchSize = %d] lr = %lf\n", DATA_BATCH, learning_rate);
    //creat CUDA streams
    hipStream_t stream[DATA_BATCH];
    for(int i=0; i<DATA_BATCH; i++){
        hipStreamCreate(&stream[i]);
    }    
    for (int iter = 0; iter < max_iter; iter ++ ){
        loss = 0;
        //double single_rate = 0;
        clock_t iter_start = clock();
        for ( int i0 = 0; i0 < batch_num; i0 ++ )
        {
            // compute the number of streams(or batch size)
            batch_size = DATA_BATCH;
            //loss = 0;
            for ( int i1 = 0; i1 < batch_size; i1 ++ )
            {
                // forward propagation
                convolution<<< FILTER_NUM, re_size, (cube_size + filter_size) * sizeof(double), stream[i1] >>>( i0 * DATA_BATCH + i1,
                                                                                                                i1, 
                                                                                                                (NEIGHBOR + 1),
                                                                                                                z,
                                                                                                                COV_LEN,
                                                                                                                FILTER_NUM,
                                                                                                                STRIDE,
                                                                                                                dataLayer.input.data_d, 
                                                                                                                conv.weights.data_d, 
                                                                                                                conv.bias.data_d, 
                                                                                                                conv.output.data_d );

                maxpooling<<< FILTER_NUM, mre_size, 0, stream[i1] >>>( i1,
                                                                       re_size,
                                                                       POOLONG_LEN,
                                                                       FILTER_NUM,
                                                                       conv.output.data_d, 
                                                                       pooling.output.data_d, 
                                                                       pooling.bias.data_d );
                
                fully_connect<<< NEU_NUM1, pooling_output_length, pooling_output_length * sizeof(double), stream[i1] >>>( i1, 
                                                                                                                          pooling_output_length,
                                                                                                                          NEU_NUM1,
                                                                                                                          pooling.output.data_d, 
                                                                                                                          fulconnect.weights.data_d, 
                                                                                                                          fulconnect.bias.data_d, 
                                                                                                                          fulconnect.output.data_d );
                
                output_and_dvalue<<< 1, NEU_NUM2, (NEU_NUM1 + NEU_NUM2) * sizeof(double), stream[i1] >>>( i0 * DATA_BATCH + i1,
                                                                                                          i1,
                                                                                                          NEU_NUM1,
                                                                                                          NEU_NUM2,
                                                                                                          true, 
                                                                                                          fulconnect.output.data_d, 
                                                                                                          out.weights.data_d, 
                                                                                                          out.bias.data_d, 
                                                                                                          out.output.data_d,
                                                                                                          dataLayer.labels.data_d,
                                                                                                          out.deltaB.data_d );
                                        

                bp_fully_connect<<<NEU_NUM1, NEU_NUM2, NEU_NUM2 * sizeof(double), stream[i1]>>>( i1, 
                                                                                          NEU_NUM1,
                                                                                          NEU_NUM2,
                                                                                          out.weights.data_d, 
                                                                                          out.deltaB.data_d, 
                                                                                          out.deltaW.data_d, 
                                                                                          fulconnect.output.data_d, 
                                                                                          fulconnect.deltaB.data_d );
                
                bp_fully_connect<<< pooling_output_length, NEU_NUM1, NEU_NUM1 * sizeof(double), stream[i1] >>>( i1,  
                                                                                                                pooling_output_length, 
                                                                                                                NEU_NUM1, 
                                                                                                                fulconnect.weights.data_d,
                                                                                                                fulconnect.deltaB.data_d, 
                                                                                                                fulconnect.deltaW.data_d,
                                                                                                                pooling.output.data_d, 
                                                                                                                //pooling.bias.data_d,
                                                                                                                pooling.deltaB.data_d );
                bp_maxpooling<<< 1, pooling_output_length, 0, stream[i1] >>>(i1,
                                                                             pooling_input_length,
                                                                             pooling_output_length,
                                                                             pooling.bias.data_d,
                                                                             pooling.deltaB.data_d,
                                                                             pooling.deltaW.data_d );

                bp_convolution<<< FILTER_NUM, filter_size, cube_size * sizeof(double), stream[i1] >>>( i0 * DATA_BATCH + i1,
                                                                                                       i1,
                                                                                                       STRIDE,
                                                                                                       (NEIGHBOR + 1),
                                                                                                       z,
                                                                                                       filter_size,
                                                                                                       FILTER_NUM,
                                                                                                       pooling_input_length,
                                                                                                       pooling.deltaW.data_d,
                                                                                                       conv.deltaW.data_d,
                                                                                                       conv.deltaB.data_d,
                                                                                                       dataLayer.input.data_d,
                                                                                                       conv.output.data_d );

            } //i1

            hipDeviceSynchronize();

            loss_function<<< 1, batch_size >>>( i0, 
                                                batch_size, 
                                                NEU_NUM2,
                                                out.output.data_d, 
                                                dataLayer.labels.data_d, 
                                                gpu_loss_values );

            checkCudaErrors(hipMemcpy(loss_values, gpu_loss_values, sizeof(double) * batch_size, hipMemcpyDeviceToHost));
            
            // hipDeviceSynchronize();
            for( int j = 0; j < batch_size; j ++ )
            {
                loss = loss + /*lossfunction(O2 + j * NEU_NUM2, processed_labels, i0 * DATA_BATCH + j);*/ loss_values[j];
            }

            //update parameters
            update_params_row<<< NEU_NUM1, NEU_NUM2 >>>( batch_size,
                                                     NEU_NUM1, 
                                                     NEU_NUM2,
                                                     learning_rate, 
                                                     out.weights.data_d,
                                                     out.deltaW.data_d,
                                                     out.bias.data_d, 
                                                     out.deltaB.data_d );

            update_params_row<<< pooling_output_length, NEU_NUM1 >>>( batch_size,
                                                            pooling_output_length,
                                                            NEU_NUM1, 
                                                            learning_rate, 
                                                            fulconnect.weights.data_d, 
                                                            fulconnect.deltaW.data_d, 
                                                            fulconnect.bias.data_d,
                                                            fulconnect.deltaB.data_d );

            update_params_col<<< FILTER_NUM, filter_size >>>( batch_size,
                                                               //FILTER_NUM,
                                                               filter_size,
                                                               FILTER_NUM,
                                                               learning_rate,
                                                               conv.weights.data_d, 
                                                               conv.deltaW.data_d,
                                                               conv.bias.data_d, 
                                                               conv.deltaB.data_d ); 
                                                               //conv.weights.data_d, 
                                                               //conv.bias.data_d );

            checkCudaErrors(hipMemset(pooling.deltaW.data_d, 0, sizeof(double) * pooling_input_length * DATA_BATCH));    
           
            //single_rate += loss;
            /*loss = loss/batch_size;
            insert_line(correct_rate, loss);//insert current loss into the line
            double new_min = *min_element(correct_rate, correct_rate + batch_size);
            if ( cur_min > new_min ) {
                cur_min = new_min;
                count = 1;
            }
            else {
                count++;
            }
            if ( count >= VALID_BATCH ) {
                learning_rate = learning_rate * decay_ratio;
                decay_ratio = decay_ratio + ra;
                if ( decay_ratio >= 1 )
                    decay_ratio = 0.999;

                fprintf(stdout,"[Cube CNN training with MBGD algo.  BatchSize = %d] lr = %lf\n",
                        DATA_BATCH, learning_rate);
                count = 1;
                cur_min = new_min;
            }
            if ( loss < MIN_ERR )
                break;*/
        } //i0

        clock_t iter_stop = clock();
        float iter_time = float(iter_stop - iter_start) / CLOCKS_PER_SEC;
        double single_rate = loss/train_size;
        logloss[iter] = single_rate;
        char str[50];
        sprintf(str, "%d", iter + 1);
        strcat(str, ",");
        fprintf(stdout,"[Cube CNN training with MBGD algo.  BatchSize = %d  Execution time: %.3f sec] Iteration %-4s loss = %lf;\n", 
                DATA_BATCH, iter_time, str, single_rate);
            
        insert_line(correct_rate, single_rate);//insert current loss into the line
        double new_min = *min_element(correct_rate, correct_rate + VALID_BATCH);
        if ( cur_min > new_min ) {
            cur_min = new_min;
            count = 1;
        }
        else {
            count++;
        }
        if ( count >= VALID_BATCH ) {
            learning_rate = learning_rate * 0.9;
            fprintf(stdout,"[Cube CNN training with MBGD algo.  BatchSize = %d] lr = %lf\n", DATA_BATCH, learning_rate);
            count = 1;
            cur_min = new_min;
        }
        if ( single_rate < MIN_ERR )
            break;
    } //iter

    fprintf(stdout,"[Cube CNN training with MBGD algo.  BatchSize = %d]", DATA_BATCH);
    end = clock();
    tt = double(end - start);
    fprintf(stdout," Completed! Global Exesution time is %.3f sec\n", tt/CLOCKS_PER_SEC);

    start = clock();
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(conv.weights.data_h, conv.weights.data_d, sizeof(double) * filter_size * FILTER_NUM, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(conv.bias.data_h, conv.bias.data_d, sizeof(double) * FILTER_NUM, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(fulconnect.bias.data_h, fulconnect.bias.data_d, sizeof(double) * NEU_NUM1, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(out.bias.data_h, out.bias.data_d, sizeof(double) * NEU_NUM2, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(fulconnect.weights.data_h, fulconnect.weights.data_d, sizeof(double) * ful_weights_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(out.weights.data_h, out.weights.data_d, sizeof(double) * out_weights_size, hipMemcpyDeviceToHost));
    
    // Write the parameters into a mat file
    MATFile * pmatFile;
    pmatFile = matOpen("model/model.mat","w");
    mxArray * m1 = mxCreateDoubleMatrix(filter_size, FILTER_NUM, mxREAL);
    memcpy((void *)mxGetPr(m1), (void *)conv.weights.data_h, sizeof(double) * filter_size * FILTER_NUM);
    matPutVariable(pmatFile, "filters", m1);

    mxArray * m2 = mxCreateDoubleMatrix(FILTER_NUM, 1, mxREAL);
    memcpy((void *)mxGetPr(m2), (void *)conv.bias.data_h, sizeof(double) * FILTER_NUM);
    matPutVariable(pmatFile, "bias0", m2);

    mxArray * m3 = mxCreateDoubleMatrix(NEU_NUM1, pooling_output_length, mxREAL);
    memcpy((void *)mxGetPr(m3), (void *)fulconnect.weights.data_h, sizeof(double) * ful_weights_size);
    matPutVariable(pmatFile, "omega1", m3);

    mxArray * m4 = mxCreateDoubleMatrix(NEU_NUM1, 1, mxREAL);
    memcpy((void *)mxGetPr(m4), (void *)fulconnect.bias.data_h, sizeof(double) * NEU_NUM1);
    matPutVariable(pmatFile, "bias1", m4);

    mxArray * m5 = mxCreateDoubleMatrix(NEU_NUM2, NEU_NUM1, mxREAL);
    memcpy((void *)mxGetPr(m5), (void *)out.weights.data_h, sizeof(double) * out_weights_size);
    matPutVariable(pmatFile, "omega2", m5);

    mxArray * m6 = mxCreateDoubleMatrix(NEU_NUM2, 1, mxREAL);
    memcpy((void *)mxGetPr(m6), (void *)out.bias.data_h, sizeof(double) * NEU_NUM2);
    matPutVariable(pmatFile, "bias2", m6);

    mxArray * m7 = mxCreateDoubleMatrix(300, 1, mxREAL);
    memcpy((void *)mxGetPr(m7), (void *)logloss, sizeof(double) * 300);
    matPutVariable(pmatFile, "loss", m7);

    matClose(pmatFile);

    delete [] logloss;
    delete [] loss_values;
    delete [] correct_rate;

    for(int i=0; i<DATA_BATCH; i++){
        hipStreamDestroy(stream[i]);
    }
    
    //test
    double right = 0;
    double accuracy_count = 0;
        dataLayer.input.data_d = gpu_processed_test;


    for ( int i1 = 0; i1 < test_size; i1 ++ ) {
        convolution<<< FILTER_NUM, re_size, (cube_size + filter_size) * sizeof(double) >>>( i1,
                                                                                            0,
                                                                                            (NEIGHBOR + 1),
                                                                                            z,
                                                                                            COV_LEN,
                                                                                            FILTER_NUM,
                                                                                            STRIDE,
                                                                                            dataLayer.input.data_d,
                                                                                            conv.weights.data_d,
                                                                                            conv.bias.data_d,
                                                                                            conv.output.data_d );
        //hipDeviceSynchronize();

        maxpooling<<< FILTER_NUM, mre_size, 0 >>>( 0,
                                                   re_size,
                                                   POOLONG_LEN,
                                                   FILTER_NUM,
                                                   conv.output.data_d, 
                                                   pooling.output.data_d, 
                                                   pooling.bias.data_d );
        //hipDeviceSynchronize();

        fully_connect<<< NEU_NUM1, pooling_output_length, pooling_output_length * sizeof(double) >>>( 0, 
                                                                                                      pooling_output_length,
                                                                                                      NEU_NUM1,
                                                                                                      pooling.output.data_d, 
                                                                                                      fulconnect.weights.data_d,
                                                                                                      fulconnect.bias.data_d,
                                                                                                      fulconnect.output.data_d );

        output_and_dvalue<<< 1, NEU_NUM2, (NEU_NUM1 + NEU_NUM2) * sizeof(double) >>>( i1,
                                                                                      0,
                                                                                      NEU_NUM1,
                                                                                      NEU_NUM2,
                                                                                      false,
                                                                                      fulconnect.output.data_d,
                                                                                      out.weights.data_d,
                                                                                      out.bias.data_d,
                                                                                      out.output.data_d,
                                                                                      NULL,
                                                                                      NULL );
        //hipDeviceSynchronize();

        checkCudaErrors(hipMemcpy(out.output.data_h, out.output.data_d, sizeof(double) * NEU_NUM2, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();

        right = count_err(test_labels, out.output.data_h, i1);
        accuracy_count = accuracy_count + right;
    }

    delete [] test_labels;


    end = clock();
    tt = float(end - start);
    fprintf(stdout, "[Cube CNN testing] Execution time is %.3f sec. ", tt/CLOCKS_PER_SEC);
  
    return accuracy_count/test_size;
}


int main(int argc, char * argv[])
{
    fprintf(stdout, "[Cube CNN training with MBGD algo] ");
      if(!InitCUDA()){
        return 0;
    }
    printf("CUDA initialized.\n");

    fprintf(stdout, "[Cube CNN training with MBGD algo] Available Device List: ");
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int device;
    for ( device = 0; device < deviceCount; ++ device )
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        if ( device == 0 )
            printf("Device %d -- %s(Default)  ", device, deviceProp.name);
        else
            printf("Device %d -- %s  ", device, deviceProp.name);
    }

    cout<<endl;

    if ( argc < 3 ) {
        fprintf(stderr, "3 input arguments required!\n");
        return 0;
    }
    int device_choosed = (int)atoi(argv[2]);
    fprintf(stdout, "[Cube CNN training with MBGD algo] Training implemented on Device %d.\n", device_choosed);
    
    hipSetDevice(device_choosed);

    double *trainset, *trainlabels;

    MATFile * datamat = matOpen(argv[1], "r");
    mxArray * train = matGetVariable(datamat,"DataSet");
    mxArray * labels = matGetVariable(datamat,"labels");

    trainset = (double*)mxGetData(train);
    trainlabels = (double*)mxGetData(labels);

    const mwSize * dim;
    dim = mxGetDimensions(train);
    matClose(datamat);

    double correct = training(trainset, trainlabels, dim[0], dim[1], dim[2]);
    fprintf(stdout,"Accuracy: %.3f%% \n", correct * 100);
    
    hipDeviceReset();
    return 0;
}
