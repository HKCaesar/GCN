#include "hip/hip_runtime.h"
#include <mat.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <matrix.h>
#include <iostream>
#include "error_util.h"
#include <hip/hip_runtime.h>
#include <algorithm>
using namespace std;

const int FILTER_NUM = 20;
const int COV_LEN = 19;
const int STRIDE = 2;
const int POOLONG_LEN = 2;
const int NEU_NUM1 = 100;
const int NEU_NUM2 = 13;
const int NEIGHBOR = 8;
float learning_rate = 0.08;
const float MIN_ERR = 0.001;
const int VALID_BATCH = 5;
const int DATA_BATCH = 10;

//Initialize CUDA
bool InitCUDA(){
	int count;
	hipGetDeviceCount(&count);
	if(count==0){
		fprintf(stderr,"There is no device.\n");
		return false;
	}
	int i;
	for (i =0; i<count;i++){
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop,i)==hipSuccess){
			if(prop.major>=1){       
                break;
			}
		}
	}
	if(i==count){
		fprintf(stderr,"There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}

template<typename T>
struct Tensor{
    int length;
    T * data_h;
    T * data_d;

    Tensor();
};

template<typename T>
Tensor<T>::Tensor()
{
    length = 0;
    data_h = NULL;
    data_d = NULL;
}

struct DataLayer{
    Tensor<float> input;
    Tensor<int> labels;

    DataLayer(int input_size, int labels_size);
    ~DataLayer();
};

DataLayer::DataLayer(int input_size, int labels_size)
{
    input.length = input_size;
    labels.length = labels_size;
}

DataLayer::~DataLayer()
{
    if(input.data_h != NULL)
        delete [] input.data_h;
    if(labels.data_h != NULL)
        delete [] labels.data_h;
    if(input.data_d != NULL)
        hipFree(input.data_d);
    if(labels.data_d != NULL)
        hipFree(labels.data_d);
}

template<typename T>
struct Layer{
    Tensor<T> input;
    Tensor<T> output;
    Tensor<T> weights;
    Tensor<T> bias;
    Tensor<T> deltaW;
    Tensor<T> deltaB;

    Layer(int input_size, int weights_size, int bias_size, int output_size, int batch_size, bool isMaxpooling, bool copyback);
    ~Layer();

private:
    void allocMemcpyCuda(int size, T ** data_h, T ** data_d, bool isMalloc, bool isCopyback);
};

template<typename T>
void Layer<T>::allocMemcpyCuda(int size, T **data_h, T **data_d, bool isMallochost, bool isInitalize)
{
    size_t sizeBytes = size * sizeof(T);
    checkCudaErrors(hipMalloc(data_d, sizeBytes));

    if (isMallochost)
    {
        *data_h = new T [size];

        if (isInitalize)
        {
            for (int i = 0; i < size; i ++)
            {
                data_h[0][i] = (2 * (rand()/T(RAND_MAX)) - 1) / 10;
            }
            checkCudaErrors(hipMemcpy(*data_d, *data_h, sizeBytes, hipMemcpyHostToDevice));
        }
    }
}

template<typename T>
Layer<T>::Layer (int input_size, int weights_size, int bias_size, int output_size, int batch_size, bool isMaxpooling, bool copyback)
{
    input.length = input_size * batch_size;
    weights.length = weights_size;
    deltaW.length = weights_size * batch_size;
    output.length = output_size * batch_size;
    bias.length = bias_size;
    deltaB.length = bias_size * batch_size;

    if ( isMaxpooling )
        bias.length = bias_size * batch_size;

    allocMemcpyCuda(input.length, &input.data_h, &input.data_d, false, false);
    allocMemcpyCuda(weights.length, &weights.data_h, &weights.data_d, true, true);
    allocMemcpyCuda(bias.length, &bias.data_h, &bias.data_d, true, true);
    allocMemcpyCuda(output.length, &output.data_h, &output.data_d, copyback, false);
    allocMemcpyCuda(deltaB.length, &deltaB.data_h, &deltaB.data_d, false, false);
    allocMemcpyCuda(deltaW.length, &deltaW.data_h, &deltaW.data_d, false, false);

    if ( isMaxpooling )
        checkCudaErrors(hipMemset(deltaW.data_d, 0, sizeof(T) * deltaW.length));
}

template<typename T>
Layer<T>::~Layer ()
{
    if (input.data_h != NULL)
        delete [] input.data_h;
    if(weights.data_h != NULL)
        delete [] weights.data_h;
    if(output.data_h != NULL)
        delete [] output.data_h;
    if(bias.data_h != NULL)
        delete [] bias.data_h;
    if(input.data_d != NULL)
        hipFree(input.data_d);
    if(output.data_d != NULL)
        hipFree(output.data_d);
    if(weights.data_d != NULL)
        hipFree(weights.data_d);
    if(bias.data_d != NULL)
        hipFree(bias.data_d);
    if(deltaW.data_d != NULL)
        hipFree(deltaW.data_d);
    if(deltaB.data_d != NULL)
        hipFree(deltaB.data_d);
}

// copy data to shared memory
__device__ void copy_data_to_shared( float * data, float * data_tmp, int tid, int offset, int head, int length )
{
	for(size_t i = tid * offset; i < (tid + 1) * offset && (i < length); i++)
    {
		data_tmp[i] = data[i + head];
	}
	__syncthreads();

}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// forward propagation kernels
// forward convolution
__global__ static void convolution( int data_id,
                                    int batch_id,
                                    int perLayerSize,
                                    int cube_len,
                                    int conv_len,
                                    int filter_num,
                                    int stride,
                                    float * input, 
                                    float * filters, 
                                    float * bias,
                                    float * output )
{

	int tid = threadIdx.x;
	int bid = blockIdx.x;
    int output_size = (cube_len - conv_len - 1) / stride + 1;

	if ( tid < output_size && bid < filter_num )
	{
        int cube_size = cube_len * perLayerSize;
		extern __shared__ float tmp[];
        int head = data_id * cube_size;
		int length = cube_size;
        int offset = (length - 1) / output_size + 1;
		copy_data_to_shared(input, tmp, tid, offset, head, length);
        
        int filterSize = conv_len * perLayerSize;
		head = bid * filterSize;
		length = filterSize;
		offset = (length - 1) / output_size + 1;
		copy_data_to_shared(filters, tmp + cube_size, tid, offset, head, length);
		__syncthreads();

		float mid = 0;
		for( int i = 0; i < filterSize; i++ ) {
            mid = mid + tmp[i + cube_size] * tmp[tid * perLayerSize * stride + i];
		}
		mid = mid + bias[bid];

        output[tid + bid * output_size + batch_id * output_size * filter_num] = 2 / (1 + (1 / exp(2 * mid))) - 1;
	}
}

// forward maxpooling
__global__ static void maxpooling( int batch_id,
                                   int input_size,
                                   int pooling_size,
                                   int group_num,
                                   float * input,
                                   float * output, 
                                   float * output_index )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
    int output_size = (input_size - 1) / pooling_size + 1;

	if ( tid < output_size && bid < group_num ) {
		float max;
		int max_index, head, tail;
		head = tid * pooling_size + bid * input_size + batch_id * input_size * group_num;
		tail = (tid + 1) * pooling_size + bid * input_size + batch_id * input_size * group_num;
		max = input[head];
		max_index = head;
		for ( int i = head; i < tail && (i < (bid + 1) * input_size + batch_id * input_size * group_num); i ++ )
        {
			if(max < input[i]){
				max = input[i];
				max_index=i;
			}
		}

		output[tid + bid * output_size + batch_id * output_size * group_num] = max;
		output_index[tid + bid * output_size + batch_id * output_size * group_num] = max_index;
	}
}

// forward fully connection
__global__ static void fully_connect( int batch_id,
                                      int input_size,
                                      int neuron_num,
                                      float * input,
                                      float * weights,
                                      float * bias,
                                      float * output )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if ( tid < input_size && bid < neuron_num )
    {
		extern __shared__ float tmp[];
		tmp[tid] = weights[bid + tid * neuron_num] * input[tid + batch_id * input_size];
		__syncthreads();

		int length = input_size;
		int offset = (length - 1) / 2 + 1;

		while ( length >= 2 )
		{
			if( tid + offset < length )
			{
				tmp[tid] = tmp[tid] + tmp[tid + offset];
			}
			offset = (offset - 1)/2 + 1;
			length = (length - 1)/2 + 1;
			__syncthreads();
		}

        if ( tid < 1 )
		    output[bid + batch_id * neuron_num] = 2/(1 + 1 / exp((tmp[0] + bias[bid]) * 2)) - 1;
	}
}

// forward output
__global__ static void output_and_dvalue( int data_id,
                                          int batch_id,
                                          int input_size,
                                          int neuron_num, 
                                          bool isBackwardPropagation,
                                          float * input, 
                                          float * weights, 
                                          float * bias, 
                                          float * output,
                                          int * labels,
                                          float * dValue )
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if ( tid < neuron_num )
    {
		// copy to shared memory
		extern __shared__ float tmp[];
		int offset = (input_size - 1) / neuron_num + 1;
		copy_data_to_shared(input, tmp, tid, offset, batch_id * input_size, input_size);
		__syncthreads();

		float mid = 0;
		for ( int i = 0; i < input_size; i ++ ) {
			mid = mid + weights[tid + i * neuron_num] * tmp[i];
		}

		tmp[tid + input_size] = exp(mid + bias[tid]);
		__syncthreads(); 

		int length = neuron_num;
		offset = (length - 1) / 2 + 1;
		while(length >= 2)
		{
			if(tid + offset < length){
				tmp[tid + input_size] = tmp[tid + input_size] + tmp[tid + input_size + offset];
			}
			offset = (offset - 1) / 2 + 1;
			length = (length - 1) / 2 + 1;
			__syncthreads();
		}

		output[tid + batch_id * neuron_num] = exp(mid + bias[tid]) / tmp[input_size];
        
        if ( isBackwardPropagation )
            dValue[tid + batch_id * neuron_num] = (output[tid + batch_id * neuron_num] - labels[tid + data_id * neuron_num]) / neuron_num;
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// backward propagation kernels
// output layer
__global__ static void bp_output( int batch_id,
                                         int input_size,
                                         int output_size, 
                                         float * weights, 
                                         float * deltaB, 
                                         float * deltaW,
                                         float * data, 
                                         float * fol_deltaZ )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if ( tid < output_size && bid < input_size ) {
		extern __shared__ float delta_A[];
		delta_A[tid] = weights[tid + bid * output_size] * deltaB[tid + batch_id * output_size];
		__syncthreads();

		deltaW[tid + bid * output_size + batch_id * input_size * output_size] = data[bid + batch_id * input_size] * deltaB[tid + batch_id * output_size]; 

		int length = output_size;
		int offset = (length - 1)/2 + 1;
		while ( length >= 2 ) 
		{
			if(tid + offset < length){
				delta_A[tid] = delta_A[tid] + delta_A[tid + offset];
			}
			length = (length - 1)/2 + 1;
			offset = (offset - 1)/2 + 1;
			__syncthreads();
		}

        if ( tid < 1 )
		    fol_deltaZ[bid + batch_id * input_size] = delta_A[0] * 
                                                      (1 + data[bid + batch_id * input_size]) * 
                                                      (1 - data[bid + batch_id * input_size]);
	}
}

// maxpooling layer
__global__ static void bp_fully_connect( int batch_id, 
                                      int input_size, 
                                      int output_size,
                                      float * weights,
                                      float * deltaB,
                                      float * deltaW,
                                      float * data,
                                      float * data_index,
                                      float * fol_deltaZ )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if( tid < output_size && bid < input_size )
    {
		extern __shared__ float mid[];
		mid[tid] = weights[tid + bid * output_size] * deltaB[tid + batch_id * output_size];
		__syncthreads();

		int length = output_size;
		int offset = (length - 1)/2 + 1;
		while ( length >= 2 ) {
			if(tid + offset < length){
				mid[tid] = mid[tid] + mid[tid+offset];
			}
			length = (length - 1)/2 + 1;
			offset = (offset - 1)/2 + 1;
			__syncthreads();
		}

        float data_tmp = data[bid + batch_id * input_size];

		deltaW[tid + bid * output_size + batch_id * input_size * output_size] = data_tmp * deltaB[tid + batch_id * output_size];
		
        if ( tid < 1 )
			fol_deltaZ[bid + batch_id * input_size] = mid[0] * (1 + data_tmp) * (1 - data_tmp);
	}
}

__global__ static void bp_maxpooling( int batch_id,
                                      int input_size,
                                      int output_size,
                                      float * bias,
                                      float * deltaB,
                                      float * fol_deltaZ )
{
    int tid = threadIdx.x;

    if ( tid < output_size )
    {
        int idx = (int)bias[tid + batch_id * output_size];

        fol_deltaZ[idx] = deltaB[tid + batch_id * output_size];
    }
}

// convolutional layer
__global__ static void bp_convolution( int data_id, 
                                       int batch_id,  
                                       int stride, 
                                       int perLayerSize,
                                       int cube_len,
                                       int filter_size,
                                       int filter_num,
                                       int output_size,
                                       float * pre_deltaB, 
                                       float * deltaW, 
                                       float * deltaB, 
                                       float * data )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if ( tid < filter_size && bid < filter_num )
	{
        int re_size = output_size / filter_num;
        int cube_size = cube_len * perLayerSize;
		int head = data_id * cube_size;
		int length = cube_size;
		int offset = (length - 1) / filter_size + 1;
		extern __shared__ float data_tmp[];
		copy_data_to_shared(data, data_tmp, tid, offset, head, length);
        __syncthreads();

		float mid0 = 0, mid1 = 0;
		for ( int i = 0; i < re_size; i ++ ) {
			mid0 = mid0 + pre_deltaB[i + bid * re_size + batch_id * output_size] * data_tmp[tid + i * perLayerSize * stride];
			mid1 = mid1 + pre_deltaB[i + bid * re_size + batch_id * output_size];
		}

		deltaW[tid + bid * filter_size + batch_id * filter_size * filter_num] = mid0 / re_size;
		
		if(tid < 1)
			deltaB[bid + batch_id * filter_num] = mid1 / re_size;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// update params kernels
// output layer
__global__ static void update_bias( int batch_size,
                                      int output_size, 
                                      float lr, 
                                      float * pre_deltaB, 
                                      float * bias )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid < batch_size && bid < output_size)
    {
		extern __shared__ float tmp[];
		tmp[tid] = pre_deltaB[bid + tid * output_size];
		__syncthreads();

		int length = batch_size;
		int offset = (length - 1)/2 + 1;
		while(length >= 2){
			if(tid + offset < length){
				tmp[tid] = tmp[tid] + tmp[tid + offset];
			}
			length = (length - 1)/2 + 1;
			offset = (offset - 1)/2 + 1;
			__syncthreads();
		}

		if(tid < 1)
			bias[bid] = bias[bid] - tmp[0] * lr ;
	}
}
// fully_connect layer
__global__ static void update_fully_connect( int batch_size, 
                                             float lr, 
                                             float * pre_weights,
                                             float * pre_deltaW,
                                             float * bias,
                                             float * pre_deltaB )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if ( tid < NEU_NUM2 && bid < NEU_NUM1 )
    {
		float mid0 = 0, mid1 = 0;
		for ( int i = 0; i < batch_size; i ++ ) {
			mid0 = mid0 + pre_deltaW[tid + bid * NEU_NUM2 + i * NEU_NUM1 * NEU_NUM2];
			mid1 = mid1 + pre_deltaB[bid + i * NEU_NUM1];
		}
		pre_weights[tid + bid * NEU_NUM2] = pre_weights[tid + bid * NEU_NUM2] - lr * mid0 ;
		
		if ( tid < 1 ) {
			bias[bid] = bias[bid] - lr * mid1 ;
		}
	}
}
// maxpooling layer
__global__ static void update_weights( int batch_size,
                                          int input_size,
                                          int output_size, 
                                          float lr, 
                                          float * pre_weights, 
                                          float * pre_deltaW )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if ( tid < output_size && bid < input_size )
    {
		float mid=0;
		for ( int i = 0; i < batch_size; i ++ )
        {
			mid = mid + pre_deltaW[tid + bid * output_size + i * input_size * output_size];
		}
		
		pre_weights[tid + bid * output_size] = pre_weights[tid + bid * output_size] - lr * mid ;
	}
}

// convolution layer
__global__ static void update_convolution( int batch_size, 
                                           int filter_size,
                                           float lr, 
                                           float * deltaW, 
                                           float * deltaB, 
                                           float * filters, 
                                           float * bias )
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if ( tid < filter_size && bid < FILTER_NUM )
	{
		float mid0 = 0, mid1 = 0;
		for ( int i = 0; i < batch_size; i ++ ) {
			mid0 = mid0 + deltaW[tid + bid * filter_size + i * filter_size * FILTER_NUM];
			mid1 = mid1 + deltaB[bid + i * FILTER_NUM];
		}

		filters[tid + bid * filter_size] = filters[tid + bid * filter_size] - lr * mid0 ;
		
		if ( tid < 1 ) {
			bias[bid] = bias[bid] - lr * mid1 ;
		}	
	}
}

__global__ static void loss_function ( int batch_id, 
                                       int batch_size, 
                                       int output_size,
                                       float * output, 
                                       int * labels, 
                                       double * loss_values)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    double sum = 0.0;
    double tmp = 0.0;
    if(tid < batch_size){
        for ( int i = 0; i < output_size; i ++ ) {

            tmp = labels[i + (batch_id * batch_size + tid) * output_size] * log(double(output[i + tid * output_size])) +
                  (1 - labels[i + (batch_id * batch_size + tid) * output_size]) * log(double(1 - output[i + tid * output_size]));

            sum = sum + tmp;
        }

        loss_values[tid] = - sum / output_size;
    }
}

//preprocessing
__global__ static void processing ( int iter, 
                                    float * data, 
                                    int * train_index, 
                                    float * processed_data, 
                                    int x, 
                                    int y, 
                                    int z, 
                                    int train_size )
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;

	if (id < train_size){
		int idx = id * (NEIGHBOR+1) * z;
		int i, j;
		for ( i = 0; i < z; i ++ ) {
			for ( j = 0; j < (NEIGHBOR + 1); j ++ ) {
				processed_data[idx] = data[train_index[j + id * (NEIGHBOR + 1)] + i * x * y];
				idx = idx + 1;	
			}
		}
	}
}

// compute correct rate
float count_err(int * test_labels, float * output, int test_idx)
{
	float right=0;
	float max =0;
	int idx = 0;
	
	for ( int i = 0; i < NEU_NUM2; i ++ )
    {
		if ( output[i] > max )
        {
			max = output[i];
			idx = i;
		}
	}
	if ( (idx + 1) == test_labels[test_idx] )
		right = 1;
	
	return right;
}

// Insert current loss value to the queue
void insert_line(float * a, float b){
	for(int i=1; i<VALID_BATCH; i++){
		a[i-1] = a[i];
	}
	a[VALID_BATCH-1] = b;
}

// shuffle
void shuffle(int * data, int * labels, int dim_row, int width){
	int index,  i;
	int temp;
	float tmp;
	srand(time(NULL));
	for(i=0; i<width; i++){
		index=rand()%(width-i) + i;
		if(index != i){
			for(int j=0; j<dim_row; j++){
				temp = data[j + i*dim_row];
				data[j + i*dim_row] = data[j +index*dim_row];
				data[j + index*dim_row] = temp;
			}

			for(int j=0; j<NEU_NUM2; j++){
				tmp = labels[j + i*NEU_NUM2];
				labels[j + i*NEU_NUM2] = labels[j + index*NEU_NUM2];
				labels[j + index*NEU_NUM2] = tmp;
			}
		}
	}
}

float training(float * data, double * labels, int x, int y, int z){
	clock_t start, end;
	start = clock();	
	float * gpu_data;//original hyperspectral image, saved in global memory
	float * gpu_processed_train;//extracted train samples
	float * gpu_processed_test;//extracted test samples
	int * gpu_train_index;//index of train samples and their neighbors
	int * gpu_test_index;//index of test samples
	int * gpu_processed_labels;//encoded train labels

	//preprocessing
	int data_size = 0;
	int * data_index = new int [x*y];
	for(int i=0; i<x*y; i++){
		if(labels[i] != 0){
			data_index[data_size]=i;
			data_size ++;
		}
	}
	int test_size = (data_size-1)/5 + 1;
	int train_size = data_size - test_size;
	fprintf(stdout,"Num of labeled sapmles : %d\n", data_size);
	int * train_index = new int [train_size * (NEIGHBOR + 1)];
	int * test_index = new int [test_size * (NEIGHBOR+1)];

	int * processed_labels = new int [train_size * NEU_NUM2]();
	int * test_labels = new int [test_size]();

	int tr=0, te=0;
	for (int i=0; i<data_size; i++){
		if (i%5 != 0){
			train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1)] = data_index[i];//index of current labeled pixel
			if(NEIGHBOR == 4)
			{
				train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) - 1] = data_index[i] - 1;
				train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) + 1] = data_index[i] + 1;
				train_index[0 + tr * (NEIGHBOR+1)] = data_index[i] - x;
				train_index[NEIGHBOR + tr * (NEIGHBOR+1)] = data_index[i] + x;
				

				if((data_index[i] % x) == 0){//first row
					train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) - 1] = train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) + 1];
				}
				if((data_index[i] % x) == (x-1)){//last row
					train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) + 1] = train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) - 1];
				}
				if((data_index[i]/x) == 0){//first column
					train_index[0 + tr * (NEIGHBOR+1)] = train_index[NEIGHBOR + tr * (NEIGHBOR+1)];
				}
				if((data_index[i]/x) == (y-1)){//last column
					train_index[NEIGHBOR + tr * (NEIGHBOR+1)] = train_index[0 + tr * (NEIGHBOR+1)];
				}
			}
			if(NEIGHBOR == 8)
			{
				train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) - 1] = data_index[i] - 1;
				train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) + 1] = data_index[i] + 1;
				for(int j0=0;j0<3;j0++){
					train_index[j0 + tr * (NEIGHBOR+1)] = data_index[i] - 1 - x + j0;
					train_index[j0+6 + tr * (NEIGHBOR+1)] = data_index[i] - 1 + x + j0;
				}

				if((data_index[i] % x) == 0){//first row
					for (int j=0; j<3; j++)
						train_index[j*3 + tr*(NEIGHBOR+1)] = train_index[j*3+2 + tr*(NEIGHBOR+1)];
				}
				if((data_index[i] % x) == (x-1)){//last row
					for(int j=0;j<3;j++)
							train_index[j*3+2 + tr*(NEIGHBOR+1)] = train_index[j*3 + tr*(NEIGHBOR+1)];
				}
				if((data_index[i]/x) == 0){//first column
					for(int j=0;j<3;j++)
						train_index[j + tr*(NEIGHBOR+1)] = train_index[j+6 + tr*(NEIGHBOR+1)];
				}
				if((data_index[i]/x) == (y-1)){//last column
					for(int j=0;j<3;j++)
						train_index[j+6  + tr*(NEIGHBOR+1)] = train_index[j + tr*(NEIGHBOR+1)];
				}
			}

			int mid = int(labels[data_index[i]]) - 1 + tr*NEU_NUM2;
			processed_labels[mid] = 1;
			tr = tr + 1;
		}
		if(i%5 == 0){
			test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1)] = data_index[i];//index of current labeled pixel
			if(NEIGHBOR == 4)
			{
				test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) - 1] = data_index[i] - 1;
				test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) + 1] = data_index[i] + 1;
				test_index[0 + te * (NEIGHBOR+1)] = data_index[i] - x;
				test_index[NEIGHBOR+ te * (NEIGHBOR+1)] = data_index[i] + x;

				if((data_index[i] % x) == 0){//first row
					test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) - 1] = test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) + 1];
				}
				if((data_index[i] % x) == (x-1)){//last row
					test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) + 1] = test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) - 1];
				}
				if((data_index[i]/x) == 0){//first column
					test_index[0 + te * (NEIGHBOR+1)] = test_index[NEIGHBOR+ te * (NEIGHBOR+1)];
				}
				if((data_index[i]/x) == (y-1)){//last column
					test_index[NEIGHBOR+ te * (NEIGHBOR+1)] = test_index[0 + te * (NEIGHBOR+1)];
				}
			}
			if(NEIGHBOR == 8)
			{
				test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) - 1] = data_index[i] - 1;
				test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) + 1] = data_index[i] + 1;
				for(int j0=0;j0<3;j0++){
					test_index[j0 + te * (NEIGHBOR+1)] = data_index[i] - 1 - x + j0;
					test_index[j0+6 + te * (NEIGHBOR+1)] = data_index[i] - 1 + x + j0;
				}

				if((data_index[i] % x) == 0){//first row
					for (int j=0; j<3; j++)
						test_index[j*3 + te*(NEIGHBOR+1)] = test_index[j*3+2 + te*(NEIGHBOR+1)];
				}
				if((data_index[i] % x) == (x-1)){//last row
					for(int j=0;j<3;j++)
						test_index[j*3+2 + te*(NEIGHBOR+1)] = test_index[j*3 + te*(NEIGHBOR+1)];
				}
				if((data_index[i]/x) == 0){//first column
					for(int j=0;j<3;j++)
						test_index[j + te*(NEIGHBOR+1)] = test_index[j+6 + te*(NEIGHBOR+1)];
				}
				if((data_index[i]/x) == (y-1)){//last column
					for(int j=0;j<3;j++)
						test_index[j+6  + te*(NEIGHBOR+1)] = test_index[j + te*(NEIGHBOR+1)];
				}
			}

			test_labels[te] = int(labels[data_index[i]]);
			te = te + 1;
		}
	}

	shuffle(train_index, processed_labels, (NEIGHBOR+1), train_size);//shuffle the samples in training set

	//malloc GPU memory, copy data to GPU
	checkCudaErrors(hipMalloc((void **) &gpu_data, sizeof(float) * x * y * z));
	checkCudaErrors(hipMemcpy(gpu_data, data, sizeof(float)* x * y * z, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void **) &gpu_train_index, sizeof(int) * train_size * (NEIGHBOR+1)));
	checkCudaErrors(hipMemcpy(gpu_train_index, train_index, sizeof(int) * train_size * (NEIGHBOR+1), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void **) &gpu_test_index, sizeof(int) * test_size * (NEIGHBOR+1)));
	checkCudaErrors(hipMemcpy(gpu_test_index, test_index, sizeof(int) * test_size * (NEIGHBOR+1), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void **) &gpu_processed_test, sizeof(float) * test_size * (NEIGHBOR+1) * z));
    checkCudaErrors(hipMalloc((void **) &gpu_processed_train, sizeof(float) * train_size * (NEIGHBOR+1) *z));

    delete [] data_index;
    delete [] train_index;
    delete [] test_index;

    int gridsize = 64;
    int blocksize = 512;
	int iter=0;

	processing<<<gridsize,blocksize>>>(iter, gpu_data, gpu_train_index, gpu_processed_train, x, y, z, train_size);
	processing<<<gridsize,blocksize>>>(iter, gpu_data, gpu_test_index, gpu_processed_test, x, y, z, test_size);

	//hipDeviceSynchronize();
	end = clock();
	float tt = float(end - start);
	fprintf(stdout,"[Samples prepared with %d Nearest-Neighbor-Pixels Strategy  Proportion of Training Samples : %d%%] Execution time : %.3f sec\n", 
            NEIGHBOR, 80, tt/CLOCKS_PER_SEC);

	checkCudaErrors(hipFree(gpu_data));
	checkCudaErrors(hipFree(gpu_train_index));
	checkCudaErrors(hipFree(gpu_test_index));
	
	// calculate length of convolutional results
	int re_size = 0;
	for ( int i = 0; i + COV_LEN < z; i += STRIDE )
    {
		re_size ++;
	}
	int mre_size = (re_size-1) / POOLONG_LEN + 1;
    int pooling_input_length = re_size * FILTER_NUM;
    int pooling_output_length = mre_size * FILTER_NUM;
	int ful_weights_size = pooling_output_length * NEU_NUM1;// Weights in full connection layer
	int out_weights_size = NEU_NUM1 * NEU_NUM2;// Weights in output layer
    int filter_size = (NEIGHBOR + 1) * COV_LEN;
    int cube_size = (NEIGHBOR + 1) * z;
	
    double * gpu_loss_values;

	// copy labels to GPU
	checkCudaErrors(hipMalloc((void**) &gpu_processed_labels, sizeof(int) * train_size * NEU_NUM2));
	checkCudaErrors(hipMemcpy(gpu_processed_labels,processed_labels,sizeof(int) * train_size * NEU_NUM2,hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &gpu_loss_values, sizeof(double) * DATA_BATCH));

    delete [] processed_labels;
	
    float loss;
    float * logloss = new float [1000]();
    double * loss_values = new double [DATA_BATCH];
	float * correct_rate = new float [VALID_BATCH];
    for(int i=0; i<VALID_BATCH; i++){
    	correct_rate[i] = 1;
    }

 	float cur_min = 1;
	int count=1;
	int batch_size = 0;
	int batch_num = train_size / DATA_BATCH;

	start = clock();

    DataLayer dataLayer(train_size * cube_size, train_size * NEU_NUM2);
    dataLayer.input.data_d = gpu_processed_train;
    dataLayer.labels.data_d = gpu_processed_labels;

    Layer<float> conv( cube_size, filter_size * FILTER_NUM, FILTER_NUM, pooling_input_length, DATA_BATCH, false, false);

    Layer<float> pooling(pooling_input_length, pooling_input_length, pooling_output_length, pooling_output_length, DATA_BATCH, true, false);
    
    Layer<float> fulconnect(pooling_output_length, pooling_output_length * NEU_NUM1, NEU_NUM1, NEU_NUM1, DATA_BATCH, false, false);

    Layer<float> out(NEU_NUM1, NEU_NUM1 * NEU_NUM2, NEU_NUM2, NEU_NUM2, DATA_BATCH, false, true);

    hipDeviceSynchronize();
    int max_iter = 300;
    fprintf(stdout, "[Cube CNN training with MBGD Algo  BatchSize = %d  Proportion of Training samples: %d%%  max_iter = %d] lr = %lf\n", DATA_BATCH, 80, max_iter, learning_rate);
	//creat CUDA streams
	hipStream_t stream[DATA_BATCH];
	for(int i=0; i<DATA_BATCH; i++){
		hipStreamCreate(&stream[i]);
	}    
	for (int iter = 0; iter < max_iter; iter ++ ) {
		loss = 0;
        clock_t iter_start = clock();
		for(int i0=0; i0<batch_num; i0++)
		{
			// compute the number of streams(or batch size)
			batch_size = DATA_BATCH;
			
			for ( int i1 = 0; i1 < batch_size; i1 ++ )
			{
				// forward propagation
                convolution<<< FILTER_NUM, re_size, (cube_size + filter_size) * sizeof(float), stream[i1] >>>( i0 * DATA_BATCH + i1,
                                                                                                                i1, 
                                                                                                                (NEIGHBOR + 1),
                                                                                                                z,
                                                                                                                COV_LEN,
                                                                                                                FILTER_NUM,
                                                                                                                STRIDE,
                                                                                                                dataLayer.input.data_d, 
                                                                                                                conv.weights.data_d, 
                                                                                                                conv.bias.data_d, 
                                                                                                                conv.output.data_d );

				maxpooling<<< FILTER_NUM, mre_size, 0, stream[i1] >>>( i1,
                                                                       re_size,
                                                                       POOLONG_LEN,
                                                                       FILTER_NUM,
                                                                       conv.output.data_d, 
                                                                       pooling.output.data_d, 
                                                                       pooling.bias.data_d );
				
				fully_connect<<< NEU_NUM1, pooling_output_length, pooling_output_length * sizeof(float), stream[i1] >>>( i1, 
                                                                                                                          pooling_output_length,
                                                                                                                          NEU_NUM1,
                                                                                                                          pooling.output.data_d, 
                                                                                                                          fulconnect.weights.data_d, 
                                                                                                                          fulconnect.bias.data_d, 
                                                                                                                          fulconnect.output.data_d );
				
				output_and_dvalue<<< 1, NEU_NUM2, (NEU_NUM1 + NEU_NUM2) * sizeof(float), stream[i1] >>>( i0 * DATA_BATCH + i1,
                                                                                                          i1,
                                                                                                          NEU_NUM1,
                                                                                                          NEU_NUM2,
                                                                                                          true, 
                                                                                                          fulconnect.output.data_d, 
                                                                                                          out.weights.data_d, 
                                                                                                          out.bias.data_d, 
                                                                                                          out.output.data_d,
                                                                                                          dataLayer.labels.data_d,
                                                                                                          out.deltaB.data_d );
										
				bp_output<<<NEU_NUM1, NEU_NUM2, NEU_NUM2 * sizeof(float), stream[i1]>>>( i1, 
                                                                                         NEU_NUM1,
                                                                                         NEU_NUM2,
                                                                                         out.weights.data_d, 
                                                                                         out.deltaB.data_d, 
                                                                                         out.deltaW.data_d, 
                                                                                         fulconnect.output.data_d, 
                                                                                         fulconnect.deltaB.data_d );
				
				bp_fully_connect<<< pooling_output_length, NEU_NUM1, NEU_NUM1 * sizeof(float), stream[i1] >>>( i1,  
                                                                                                               pooling_output_length, 
                                                                                                               NEU_NUM1, 
                                                                                                               fulconnect.weights.data_d,
                                                                                                               fulconnect.deltaB.data_d, 
                                                                                                               fulconnect.deltaW.data_d,
                                                                                                               pooling.output.data_d, 
                                                                                                               pooling.bias.data_d,
                                                                                                               pooling.deltaB.data_d );
										     
				bp_maxpooling<<< 1, pooling_output_length, 0, stream[i1] >>>(i1,
                                                                             pooling_input_length,
                                                                             pooling_output_length,
                                                                             pooling.bias.data_d,
                                                                             pooling.deltaB.data_d,
                                                                             pooling.deltaW.data_d );

				bp_convolution<<< FILTER_NUM, filter_size, cube_size * sizeof(float), stream[i1] >>>( i0 * DATA_BATCH + i1,
                                                                                                       i1,
                                                                                                       STRIDE,
                                                                                                       (NEIGHBOR + 1),
                                                                                                       z,
                                                                                                       filter_size,
                                                                                                       FILTER_NUM,
                                                                                                       pooling_input_length,
                                                                                                       pooling.deltaW.data_d,
                                                                                                       conv.deltaW.data_d,
                                                                                                       conv.deltaB.data_d,
                                                                                                       dataLayer.input.data_d );

			}

            hipDeviceSynchronize();

            loss_function<<< 1, batch_size >>>( i0, 
                                                batch_size, 
                                                NEU_NUM2,
                                                out.output.data_d, 
                                                dataLayer.labels.data_d, 
                                                gpu_loss_values );

            checkCudaErrors(hipMemcpy(loss_values, gpu_loss_values, sizeof(double) * batch_size, hipMemcpyDeviceToHost));
			
            hipDeviceSynchronize();
			for( int j = 0; j < batch_size; j ++ )
            {
                if ( isnan(loss_values[j]) )
                    loss_values[j] = 0.0001;

				loss = loss + loss_values[j];
			}

			//update parameters
			update_bias<<< NEU_NUM2, batch_size, sizeof(float) * batch_size >>>( batch_size, 
                                                                                    NEU_NUM2,
                                                                                    learning_rate, 
                                                                                    out.deltaB.data_d, 
                                                                                    out.bias.data_d );
			
			update_fully_connect<<< NEU_NUM1, NEU_NUM2 >>>( batch_size, 
                                                            learning_rate, 
                                                            out.weights.data_d, 
                                                            out.deltaW.data_d, 
                                                            fulconnect.bias.data_d,
                                                            fulconnect.deltaB.data_d );
			
			update_weights<<< pooling_output_length, NEU_NUM1 >>>( batch_size,
                                                                      pooling_output_length, 
                                                                      NEU_NUM1, 
                                                                      learning_rate, 
                                                                      fulconnect.weights.data_d, 
                                                                      fulconnect.deltaW.data_d );
			
            update_convolution<<< FILTER_NUM, filter_size >>>( batch_size, 
                                                               filter_size,
                                                               learning_rate, 
                                                               conv.deltaW.data_d, 
                                                               conv.deltaB.data_d, 
                                                               conv.weights.data_d, 
                                                               conv.bias.data_d );
	
            checkCudaErrors(hipMemset(pooling.deltaW.data_d, 0, sizeof(float) * pooling_input_length * DATA_BATCH));    
        } //i0

        clock_t iter_stop = clock();
        float iter_time = float(iter_stop - iter_start) / CLOCKS_PER_SEC;
		double single_rate = loss/train_size;
       		logloss[iter] = single_rate;

		
		fprintf(stdout,"[Cube CNN training with MBGD Algo  BatchSize = %d  Proportion of Training Samples: %d%%  max_iter = %d  Execution time: %.3f sec] Epoch %d, loss = %lf;\n", 
                DATA_BATCH, 80, max_iter,  iter_time, iter + 1, single_rate);

        	
		insert_line(correct_rate, single_rate);//insert current loss into the line
		float new_min = *min_element(correct_rate, correct_rate + VALID_BATCH);
        	if(cur_min > new_min){
            		cur_min = new_min;
		     	count = 1;
        	}
        	else{
            		count++;
        	}
        	if(count >= VALID_BATCH ) {
            		learning_rate = learning_rate * 0.9;
            		fprintf(stdout,"[Cube CNN training with MBGD Algo  BatchSize = %d  Proportion of Training Samples: %d%%  max_iter = %d] lr = %lf\n",
                            DATA_BATCH, 80, max_iter, learning_rate);

            		count = 1;
            		cur_min = new_min;
        	}
        	if(single_rate < MIN_ERR)
            		break;
	} // iter

	fprintf(stdout,"[Cube CNN training with MBGD Algo  BatchSize = %d  Proportion of Training Samples: %d%%  max_iter = %d ]", DATA_BATCH, 80, max_iter);
	end = clock();
	tt = float(end - start);
	fprintf(stdout," Completed! Global Exesution time is %.3f sec\n", tt/CLOCKS_PER_SEC);

	start = clock();
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy(conv.weights.data_h, conv.weights.data_d, sizeof(float) * filter_size * FILTER_NUM, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(conv.bias.data_h, conv.bias.data_d, sizeof(float) * FILTER_NUM, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(fulconnect.bias.data_h, fulconnect.bias.data_d, sizeof(float) * NEU_NUM1, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(out.bias.data_h, out.bias.data_d, sizeof(float) * NEU_NUM2, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(fulconnect.weights.data_h, fulconnect.weights.data_d, sizeof(float) * ful_weights_size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(out.weights.data_h, out.weights.data_d, sizeof(float) * out_weights_size, hipMemcpyDeviceToHost));
	
	// Write the parameters into a mat file
	MATFile * pmatFile;
	pmatFile = matOpen("model/model.mat","w");
	mxArray * m1 = mxCreateDoubleMatrix(filter_size, FILTER_NUM, mxREAL);
	memcpy((void *)mxGetPr(m1), (void *)conv.weights.data_h, sizeof(float) * filter_size * FILTER_NUM);
	matPutVariable(pmatFile, "filters", m1);

	mxArray * m2 = mxCreateDoubleMatrix(FILTER_NUM, 1, mxREAL);
	memcpy((void *)mxGetPr(m2), (void *)conv.bias.data_h, sizeof(float) * FILTER_NUM);
	matPutVariable(pmatFile, "bias0", m2);

	mxArray * m3 = mxCreateDoubleMatrix(NEU_NUM1, pooling_output_length, mxREAL);
	memcpy((void *)mxGetPr(m3), (void *)fulconnect.weights.data_h, sizeof(float) * ful_weights_size);
	matPutVariable(pmatFile, "omega1", m3);

	mxArray * m4 = mxCreateDoubleMatrix(NEU_NUM1, 1, mxREAL);
    memcpy((void *)mxGetPr(m4), (void *)fulconnect.bias.data_h, sizeof(float) * NEU_NUM1);
	matPutVariable(pmatFile, "bias1", m4);

	mxArray * m5 = mxCreateDoubleMatrix(NEU_NUM2, NEU_NUM1, mxREAL);
	memcpy((void *)mxGetPr(m5), (void *)out.weights.data_h, sizeof(float) * out_weights_size);
	matPutVariable(pmatFile, "omega2", m5);

	mxArray * m6 = mxCreateDoubleMatrix(NEU_NUM2, 1, mxREAL);
	memcpy((void *)mxGetPr(m6), (void *)out.bias.data_h, sizeof(float) * NEU_NUM2);
	matPutVariable(pmatFile, "bias2", m6);

    mxArray * m7 = mxCreateDoubleMatrix(300, 1, mxREAL);
    memcpy((void *)mxGetPr(m7), (void *)logloss, sizeof(float) * 300);
    matPutVariable(pmatFile, "loss", m7);

	matClose(pmatFile);

    delete [] logloss;
    delete [] loss_values;
    delete [] correct_rate;

	for(int i=0; i<DATA_BATCH; i++){
		hipStreamDestroy(stream[i]);
	}
	
	//test
	float right = 0;
	float accuracy_count = 0;
        dataLayer.input.data_d = gpu_processed_test;


	for (int i1=0; i1<test_size; i1++){
		convolution<<< FILTER_NUM, re_size, (cube_size + filter_size) * sizeof(float)/*, testStream[i1]*/ >>>( i1,
                                                                                                                0,
                                                                                                                (NEIGHBOR + 1),
                                                                                                                z,
                                                                                                                COV_LEN,
                                                                                                                FILTER_NUM,
                                                                                                                STRIDE,
                                                                                                                dataLayer.input.data_d,
                                                                                                                conv.weights.data_d,
                                                                                                                conv.bias.data_d,
                                                                                                                conv.output.data_d );
		//hipDeviceSynchronize();

		maxpooling<<< FILTER_NUM, mre_size, 0/*, testStream[i1]*/ >>>( 0,
                                                                       re_size,
                                                                       POOLONG_LEN,
                                                                       FILTER_NUM,
                                                                       conv.output.data_d, 
                                                                       pooling.output.data_d, 
                                                                       pooling.bias.data_d );
		//hipDeviceSynchronize();

		fully_connect<<< NEU_NUM1, pooling_output_length, pooling_output_length * sizeof(float)/*, testStream[i1]*/ >>>( 0, 
                                                                                                                      pooling_output_length,
                                                                                                                      NEU_NUM1,
                                                                                                                      pooling.output.data_d, 
                                                                                                                      fulconnect.weights.data_d,
                                                                                                                      fulconnect.bias.data_d,
                                                                                                                      fulconnect.output.data_d );

		output_and_dvalue<<< 1, NEU_NUM2, (NEU_NUM1 + NEU_NUM2) * sizeof(float)/*, testStream[i1]*/ >>>( i1,
                                                                                                          0,
                                                                                                          NEU_NUM1,
                                                                                                          NEU_NUM2,
                                                                                                          false,
                                                                                                          fulconnect.output.data_d,
                                                                                                          out.weights.data_d,
                                                                                                          out.bias.data_d,
                                                                                                          out.output.data_d,
                                                                                                          NULL,
                                                                                                          NULL );
		//hipDeviceSynchronize();

		checkCudaErrors(hipMemcpy(out.output.data_h, out.output.data_d, sizeof(float) * NEU_NUM2, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();

		right = count_err(test_labels, out.output.data_h, i1);
		accuracy_count = accuracy_count + right;
	}

    delete [] test_labels;


	end = clock();
	tt = float(end - start);
	fprintf(stdout, "[Cube CNN testing] Execution time is %.3fs. ", tt/CLOCKS_PER_SEC);
  
    return accuracy_count/test_size;
}


int main(int argc, char * argv[])
{
    fprintf(stdout, "[Cube CNN training with MBGD Algorithm] ");
  	if(!InitCUDA()){
		return 0;
	}
	printf("CUDA initialized.\n");

    fprintf(stdout, "[Cube CNN training with MBGD Algorithm] Available Device List: ");
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int device;
    for (device = 0; device < deviceCount; ++ device)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        if (device == 0)
            printf("Device %d -- %s(Default)  ", device, deviceProp.name);
        else
            printf("Device %d -- %s  ", device, deviceProp.name);
    }

    cout<<endl;
    int device_choosed = 1;
    fprintf(stdout, "[Cube CNN training with MBGD Algo] Training implemented on Device %d.\n", device_choosed);
    hipSetDevice(1);

	float *trainset;
    double *trainlabels;
	if(argc!=2){
		fprintf(stderr, "2 input arguments required!");
		return 0;
	}

	MATFile * datamat = matOpen(argv[1], "r");
	mxArray * train = matGetVariable(datamat,"DataSet");
	mxArray * labels = matGetVariable(datamat,"labels");

	trainset = (float*)mxGetData(train);
	trainlabels = (double*)mxGetData(labels);

	const mwSize * dim;
	dim = mxGetDimensions(train);
	matClose(datamat);

	float correct = training(trainset, trainlabels, dim[0], dim[1], dim[2]);
	fprintf(stdout,"Accuracy: %.3f%% \n", correct * 100);
    
    hipDeviceReset();
	return 0;
}
